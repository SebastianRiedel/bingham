#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "include/bingham/cuda_wrapper.h"
#include "hiprand.h"
#include "bingham/olf.h"

#include <math.h>

#define MAX(x,y) ((x) > (y) ? (x) : (y))
#define MIN(x,y) ((x) < (y) ? (x) : (y))

#define cu_malloc(x, sz, msg) do{ if (hipMalloc(x, sz) != hipSuccess) printf(msg); } while (0)
#define cu_free(x, msg) do{ if (hipFree(x) != hipSuccess) printf(msg); } while (0)

hiprandGenerator_t gen;

//#define CUDA_LAUNCH_BLOCKING 1

__device__ __constant__ int big_primes[100] = {996311, 163573, 481123, 187219, 963323, 103769, 786979, 826363, 874891, 168991, 442501, 318679, 810377, 471073, 914519, 251059, 321983, 220009, 211877, 875339, 605603, 578483, 219619, 860089, 644911, 398819, 544927, 444043, 161717, 301447, 201329, 252731, 301463, 458207, 140053, 906713, 946487, 524389, 522857, 387151, 904283, 415213, 191047, 791543, 433337, 302989, 445853, 178859, 208499, 943589, 957331, 601291, 148439, 296801, 400657, 829637, 112337, 134707, 240047, 669667, 746287, 668243, 488329, 575611, 350219, 758449, 257053, 704287, 252283, 414539, 647771, 791201, 166031, 931313, 787021, 520529, 474667, 484361, 358907, 540271, 542251, 825829, 804709, 664843, 423347, 820367, 562577, 398347, 940349, 880603, 578267, 644783, 611833, 273001, 354329, 506101, 292837, 851017, 262103, 288989};

__device__ __constant__ double b_SR[3] = {0.2878,    -5.6214,      7.7247};
__device__ __constant__ double b_SN[3] = {0.1521,    -7.1290,     10.7090};
__device__ __constant__ double b_SL[3] = {0.2238,    -5.1827,      6.8242};
__device__ __constant__ double b_SA[3] = {0.1618,    -6.3992,      8.0207};
__device__ __constant__ double b_SB[3] = {0.2313,    -6.3463,      8.0651};

__device__ __constant__ double b_ER[3] = {0.3036,     0.2607,   -125.8843};
__device__ __constant__ double b_EN[3] = {0.1246,     1.4406,   -185.8350};
__device__ __constant__ double b_EL[3] = {0.2461,     0.2624,   -140.0192};
__device__ __constant__ double b_EA[3] = {0.1494,     0.2114,   -139.4324};
__device__ __constant__ double b_EB[3] = {0.2165,     0.2600,   -135.5203};

__device__ inline double cu_sigmoid(double x, const double *b)
{
  return b[0] + (1 - b[0]) / (1 + exp(-b[1]-b[2]*x));
}

__device__ inline double cu_logistic(double x, double *b)
{
  return 1.0 / (1.0 + exp(-x*b[1]-b[0]));
}

void copy_double_matrix_to_gpu(cu_double_matrix_t *dev_dest, double **host_src, int n, int m) {
  dev_dest->n = n;
  dev_dest->m = m;
  if (hipMalloc(&(dev_dest->ptr), m*n*sizeof(double)) != hipSuccess) {
    printf("double 2d malloc\n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0], n * m * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
    printf("double 2d copy\n");
  }      
} 

void copy_int_matrix_to_gpu(cu_int_matrix_t *dev_dest, int **host_src, int n, int m) {
  dev_dest->n = n;
  dev_dest->m = m;
  if (hipMalloc(&(dev_dest->ptr), m*n*sizeof(int)) != hipSuccess) {
    printf("int 2d malloc \n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0], n * m * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
    printf("int 2d copy\n");
  }      
}

void copy_double_matrix3d_to_gpu(cu_double_matrix3d_t *dev_dest, double ***host_src, int n, int m, int p) {
  dev_dest->n = n; dev_dest->m = m; dev_dest->p = p;
  if (hipMalloc(&(dev_dest->ptr), n * m * p * sizeof(double)) != hipSuccess) {
    printf("3d malloc\n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0][0], n * m * p * sizeof(double), hipMemcpyHostToDevice)) {
    printf("3d copy\n");
  }
}

void copy_double_arr_to_gpu(cu_double_arr_t *dev_dest, double *host_src, int n) {
  dev_dest->n = n;
  if (hipMalloc(&(dev_dest->ptr), n * sizeof(double)) != hipSuccess) {
    printf("double arr malloc\n");
  }
  if (hipMemcpy(dev_dest->ptr, host_src, n * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
    printf("double arr copy\n");
  }
}

void copy_int_arr_to_gpu(cu_int_arr_t *dev_dest, int *host_src, int n) {
  dev_dest->n = n;
  if (hipMalloc(&(dev_dest->ptr), n * sizeof(int)) != hipSuccess) {
    printf("int arr malloc\n");
  }
  if (hipMemcpy(dev_dest->ptr, host_src, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
    printf("int arr copy\n");
  }
}

__device__ void cu_quaternion_to_rotation_matrix(double R[][3], double q[]) {
  double a = q[0];
  double b = q[1];
  double c = q[2];
  double d = q[3];

  R[0][0] = a*a + b*b - c*c - d*d;
  R[0][1] = 2*b*c - 2*a*d;
  R[0][2] = 2*b*d + 2*a*c;
  R[1][0] = 2*b*c + 2*a*d;
  R[1][1] = a*a - b*b + c*c - d*d;
  R[1][2] = 2*c*d - 2*a*b;
  R[2][0] = 2*b*d - 2*a*c;
  R[2][1] = 2*c*d + 2*a*b;
  R[2][2] = a*a - b*b - c*c + d*d;
}

__device__ double cu_dot(double x[], double y[], int n) {
  int i;
  double z = 0.0;
  for (i = 0; i < n; i++)
    z += x[i]*y[i];
  return z;
}

__device__ void cu_matrix_vec_mult_3(double *y, double A[][3], double *x, int n) {
  int i;
  if (y == x) { // dbug
    printf("**************FIX CU_MATRIX_VEC_MULT CALL!\n");
  }
  for (i = 0; i < n; i++)
    y[i] = cu_dot(A[i], x, 3);
}

// adds two vectors, z = x+y
__device__ void cu_add(double z[], double x[], double y[], int n) {
  int i;
  for (i = 0; i < n; i++)
    z[i] = x[i] + y[i];
}

__device__ double cu_norm(double x[], int n) {
  double d = 0.0;
  int i;

  for (i = 0; i < n; i++)
    d += x[i]*x[i];

  return sqrt(d);
}

__device__ void cu_normalize(double y[], double x[], int n) {
  double d = cu_norm(x, n);
  int i;
  for (i = 0; i < n; i++)
    y[i] = x[i]/d;
}

// compute the pdf of a normal random variable
__device__ double cu_normpdf(double x, double mu, double sigma) {
  double dx = x - mu;

  return exp(-dx*dx / (2*sigma*sigma)) / (sqrt(2*M_PI) * sigma);
}

// invert a quaternion
__device__ void cu_quaternion_inverse(double q_inv[4], double *q) {
  q_inv[0] = q[0];
  q_inv[1] = -q[1];
  q_inv[2] = -q[2];
  q_inv[3] = -q[3];
}

// multiplies a vector by a scalar, y = c*x
__device__ void cu_mult(double y[], double x[], double c, int n)
{
  int i;
  for (i = 0; i < n; i++)
    y[i] = c*x[i];
}

// computes the max of x
int arr_max_i(int *x, int n)
{
  int i;

  int y = x[0];
  for (i = 1; i < n; i++)
    if (x[i] > y)
      y = x[i];

  return y;
}

void cu_init() {
  hipError_t err = hipInit(0);
  //if (err != 0) 
  printf("Init error: %d\n", err);
}

void cu_init_model(scope_model_data_t *model_data, cu_model_data_t *cu_model) {
  // Allocate all the memory
  copy_double_matrix_to_gpu(&(cu_model->points), model_data->pcd_model->points, model_data->pcd_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->normals), model_data->pcd_model->normals, model_data->pcd_model->num_points, 3);
  copy_double_arr_to_gpu(&(cu_model->normalvar), model_data->pcd_model->normalvar, model_data->pcd_model->num_points);
  copy_double_matrix_to_gpu(&(cu_model->lab), model_data->pcd_model->lab, model_data->pcd_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->ved), model_data->pcd_model->ved, model_data->pcd_model->num_points, 66);
  /*copy_double_matrix_to_gpu(&(cu_model->color_avg_cov), model_data->color_model->avg_cov, 3, 3);
  copy_int_arr_to_gpu(&(cu_model->color_cnts1), model_data->color_model->cnts[0], model_data->color_model->num_points);
  copy_int_arr_to_gpu(&(cu_model->color_cnts2), model_data->color_model->cnts[1], model_data->color_model->num_points);
  copy_double_matrix_to_gpu(&(cu_model->color_means1), model_data->color_model->means[0], model_data->color_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->color_means2), model_data->color_model->means[1], model_data->color_model->num_points, 3);
  copy_double_matrix3d_to_gpu(&(cu_model->color_cov1), model_data->color_model->covs[0], model_data->color_model->num_points, 3, 3);
  copy_double_matrix3d_to_gpu(&(cu_model->color_cov2), model_data->color_model->covs[1], model_data->color_model->num_points, 3, 3);*/
  //copy_double_matrix_to_gpu(&(cu_model->fpfh_shapes), model_data->fpfh_model->shapes, model_data->fpfh_model->shape_length, 33);
  copy_double_matrix_to_gpu(&(cu_model->range_edges_model_views), model_data->range_edges_model->views, model_data->range_edges_model->num_views, 3);
  copy_int_arr_to_gpu(&(cu_model->range_edges_view_idx), model_data->range_edges_model->view_idx, model_data->range_edges_model->num_views);
  copy_int_arr_to_gpu(&(cu_model->range_edges_view_cnt), model_data->range_edges_model->view_cnt, model_data->range_edges_model->num_views);
  copy_double_matrix_to_gpu(&(cu_model->range_edges_points), model_data->range_edges_model->pcd->points, model_data->range_edges_model->pcd->num_points, 3);

  hipMalloc(&(cu_model->score_comp_models), sizeof(score_comp_models_t));
  hipMemcpy(cu_model->score_comp_models, model_data->score_comp_models, sizeof(score_comp_models_t), hipMemcpyHostToDevice);
  //memcpy(&cu_model->score_comp_models, model_data->score_comp_models, sizeof(score_comp_models_t));
    
  cu_model->num_points = model_data->pcd_model->num_points;
  cu_model->num_views = model_data->range_edges_model->num_views;
  int n_edge = arr_max_i(model_data->range_edges_model->view_cnt, model_data->range_edges_model->num_views);
  cu_model->max_num_edges = n_edge;
}

void cu_init_obs(scope_obs_data_t *obs_data, cu_obs_data_t *cu_obs, scope_params_t *params) {

  copy_double_matrix_to_gpu(&(cu_obs->range_image), obs_data->obs_range_image->image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_cnt), obs_data->obs_range_image->cnt, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_points), obs_data->obs_range_image->points, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_normals), obs_data->obs_range_image->normals, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  if (params->use_colors)
    copy_double_matrix3d_to_gpu(&(cu_obs->obs_lab_image), obs_data->obs_lab_image, 3, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_idx), obs_data->obs_range_image->idx, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix_to_gpu(&(cu_obs->range_image_pcd_obs_lab), obs_data->pcd_obs->lab, obs_data->pcd_obs->num_points, 3);
  //copy_double_matrix_to_gpu(&(cu_obs->pcd_obs_fpfh), obs_data->pcd_obs->fpfh, obs_data->pcd_obs->fpfh_length, 33);
  copy_double_matrix_to_gpu(&(cu_obs->edge_image), obs_data->obs_edge_image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix_to_gpu(&(cu_obs->segment_affinities), obs_data->obs_segment_affinities, obs_data->num_obs_segments, obs_data->num_obs_segments);

  cu_obs->range_image_data.res = obs_data->obs_range_image->res;
  cu_obs->range_image_data.min0 = obs_data->obs_range_image->min[0];
  cu_obs->range_image_data.min1 = obs_data->obs_range_image->min[1];
  cu_obs->range_image_data.w = obs_data->obs_range_image->w;
  cu_obs->range_image_data.h = obs_data->obs_range_image->h;
  cu_obs->num_obs_segments = obs_data->num_obs_segments;

  // CONTINUE HERE FOR OBS DATA COPYING ********************************
}



void cu_free_all_the_model_things(cu_model_data_t *cu_model) {
  hipFree(cu_model->points.ptr);
  hipFree(cu_model->normals.ptr);
  hipFree(cu_model->normalvar.ptr);
  hipFree(cu_model->lab.ptr);
  hipFree(cu_model->ved.ptr);
  /*hipFree(cu_model->color_avg_cov.ptr);
  hipFree(cu_model->color_means1.ptr);
  hipFree(cu_model->color_means2.ptr);
  hipFree(cu_model->color_cov1.ptr);
  hipFree(cu_model->color_cov2.ptr);
  hipFree(cu_model->color_cnts1.ptr);
  hipFree(cu_model->color_cnts2.ptr);*/
  //hipFree(cu_model->fpfh_shapes.ptr);
  hipFree(cu_model->range_edges_model_views.ptr);
  hipFree(cu_model->range_edges_points.ptr);
  hipFree(cu_model->range_edges_view_idx.ptr);
  hipFree(cu_model->range_edges_view_cnt.ptr);
  hipFree(cu_model->score_comp_models);
}

void cu_free_all_the_obs_things(cu_obs_data_t *cu_obs, scope_params_t *params) {
  hipFree(cu_obs->range_image.ptr);
  hipFree(cu_obs->range_image_idx.ptr);
  hipFree(cu_obs->range_image_pcd_obs_lab.ptr);
  //hipFree(cu_obs->pcd_obs_fpfh.ptr);
  hipFree(cu_obs->edge_image.ptr);
  hipFree(cu_obs->range_image_points.ptr);
  hipFree(cu_obs->range_image_normals.ptr);
  hipFree(cu_obs->range_image_cnt.ptr);
  if (params->use_colors)
    hipFree(cu_obs->obs_lab_image.ptr);
  hipFree(cu_obs->segment_affinities.ptr);
}

void cu_free_all_the_things(cu_model_data_t *cu_model, cu_obs_data_t *cu_obs, scope_params_t *cu_params, scope_params_t *params) {
  // Free ALL the things!!!
  cu_free(cu_params, "params");
  hipError_t cudaerr = hipGetLastError();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

  cu_free_all_the_model_things(cu_model);
  cu_free_all_the_obs_things(cu_obs, params);
  hiprandDestroyGenerator(gen);
}

void cu_free_all_the_things_mope(cu_model_data_t cu_model[], cu_obs_data_t *cu_obs, int num_models, scope_params_t *params) {
  // Free ALL the things!!!
  
  for (int i = 0; i < num_models; ++i) {
    cu_free_all_the_model_things(&cu_model[i]);

  }
  cu_free_all_the_obs_things(cu_obs, params);
  hiprandDestroyGenerator(gen);
}

void cu_init_scoring(scope_model_data_t *model_data, scope_obs_data_t *obs_data, cu_model_data_t *cu_model, cu_obs_data_t *cu_obs, scope_params_t **cu_params, scope_params_t *params) {

  cu_malloc(cu_params, sizeof(scope_params_t), "params");
  hipMemcpy(*cu_params, params, sizeof(scope_params_t), hipMemcpyHostToDevice);

  cu_init_model(model_data, cu_model);
  cu_init_obs(obs_data, cu_obs, params);

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
}

void cu_init_scoring_mope(scope_model_data_t model_data[], scope_obs_data_t *obs_data, int num_models, cu_model_data_t cu_model[], cu_obs_data_t *cu_obs, scope_params_t *params) {
  
  // Allocate all the memory
  for (int i = 0; i < num_models; ++i) {
    cu_init_model(&model_data[i], &cu_model[i]);
  }
  cu_init_obs(obs_data, cu_obs, params);
  
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
}

__device__ void cu_range_image_xyz2sub(int *i, int *j, cu_range_image_data_t range_image, double xyz[])
{
  //TODO: use range image viewpoint

  double d = cu_norm(xyz, 3);
  double x = atan2(xyz[0], xyz[2]);
  double y = acos(xyz[1] / d);

  int cx = (int)floor((x - range_image.min0) / range_image.res);
  int cy = (int)floor((y - range_image.min1) / range_image.res);

  *i = cx;
  *j = cy;

  if (!((cx >= 0 && cy>=0) && (cx < range_image.w) && (cy < range_image.h))) {
    *i = -1;
    *j = -1;
  }
}

/*                                                                                                                                                                                                                 
 * compute viewpoint (in model coordinates) for model placement (x,q) assuming observed viewpoint = (0,0,0)
 */
__device__ void cu_model_pose_to_viewpoint(double *vp, double *x, double *q)
{
  double q_inv[4];
  cu_quaternion_inverse(q_inv, q);
  double R_inv[3][3];
  cu_quaternion_to_rotation_matrix(R_inv,q_inv);
  cu_matrix_vec_mult_3(vp, R_inv, x, 3);
  cu_mult(vp, vp, -1, 3);
}

__global__ void cu_add_matrix_rows_slow(double *out_array, double *in_matrix, int n, int m, int *m_arr) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i >= n)
    return;

  int limit = m;
  if (m_arr)
    limit = m_arr[i];
  
  out_array[i] = 0.0;
  for (int j = 0; j < limit; ++j) {
    out_array[i] += in_matrix[j + i * m];
  }
}

__global__ void cu_add_matrix_rows_medium(double *out_array, double *in_matrix, int n, int m, int *m_arr) {
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i >= n || j >= m)
    return;

  int limit = m;
  if (m_arr)
    limit = m_arr[i];

  extern __shared__ double tmps[];
  tmps[threadIdx.x] = 0.0;
  for (int k = j; k < limit; k += blockDim.x) {
    tmps[threadIdx.x] += in_matrix[k + i * m];
  }

  __syncthreads();
  
  if (j == 0) {
    out_array[i] = 0.0;
    for (int k = 0; k < blockDim.x; ++k) {
      out_array[i] += tmps[k];
    }
  }
}

__global__ void cu_divide_matrix_with_vector(double *out_matrix, double *in_matrix, double *scaling_array, int n, int m, int *m_arr) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= n || j >= m)
    return;
  if (m_arr && j >= m_arr[i])
    return;  

  out_matrix[j + i * m] = in_matrix[j + i * m] / scaling_array[i];
}
    
__global__ void cu_get_validation_points(int *idx, int total_pts, int needed, int num_samples, uint *rands)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= needed || i >= num_samples)
    return;

  if (needed == total_pts) {  // use all the points
    idx[j + i * needed] = j;
  } else {
    idx[j + i * needed] = ((rands[(i << 1)] % total_pts) + (j * (big_primes[rands[(i << 1) + 1] % 100] % total_pts))) % total_pts;
  }
}

__global__ void cu_get_sub_cloud_at_pose(double *cloud, cu_double_matrix_t points, double *x, double *q, int *idx, int num_samples, int n)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= n || i >= num_samples)
    return;

  int i_arr = j + i * n;

  double R[3][3];
  cu_quaternion_to_rotation_matrix(R, &q[i * 4]);
  double dest[3]; // In local memory so we access global memory less
  dest[0] = points.ptr[idx[i_arr] * points.m]; 
  dest[1] = points.ptr[idx[i_arr] * points.m + 1]; 
  dest[2] = points.ptr[idx[i_arr] * points.m + 2];
  double tmp[3];
  cu_matrix_vec_mult_3(tmp, R, dest, 3);
  cu_add(dest, tmp, &x[i * 3], 3);
  cloud[3 * i_arr] = dest[0]; cloud[3*i_arr + 1] = dest[1]; cloud[3*i_arr + 2] = dest[2];
}

__global__ void cu_get_sub_cloud_normals_rotated(double *cloud_normals, cu_double_matrix_t normals, double *q, int *idx, int num_samples, int n)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= n || i >= num_samples)
    return;

  int i_arr = j + i * n;

  double R[3][3];
  cu_quaternion_to_rotation_matrix(R, &q[i * 4]);
  double *row;
  double dest[3];

  row = &normals.ptr[idx[i_arr] * normals.m];

  double tmp[3];
  tmp[0] = row[0]; tmp[1] = row[1]; tmp[2] = row[2];
  cu_matrix_vec_mult_3(dest, R, tmp, 3);
  cloud_normals[3*i_arr] = dest[0]; cloud_normals[3*i_arr+1] = dest[1]; cloud_normals[3*i_arr + 2] = dest[2];
}
__global__ void cu_populate_xi_yi(int *xi, int *yi, double *cloud, cu_range_image_data_t range_image_data, int num_samples, int n, int *n_arr) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (i >= num_samples || j >= n)
    return;
  if (n_arr && j >= n_arr[i])
    return;

  int i_arr = j + i * n;
 
  double dest[3];
  dest[0] = cloud[3*i_arr]; 
  dest[1] = cloud[3*i_arr + 1]; 
  dest[2] = cloud[3*i_arr + 2];
  cu_range_image_xyz2sub(&xi[i_arr], &yi[i_arr], range_image_data, dest);
  if (0)
    printf("%d %d %d\n", i_arr, xi[i_arr], yi[i_arr]);
}

__global__ void cu_compute_visibility_prob(double *cu_vis_prob, double *cu_cloud, double *cu_normals, int *cu_xi, int *cu_yi, cu_range_image_data_t ri_data, 
					   cu_double_matrix_t range_image, double vis_thresh, int search_radius, int num_samples, int n, int *n_arr) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= n)
    return;
  if (n_arr && j >= n_arr[i])
    return;

  int i_arr = j + i * n;

  int xi = cu_xi[i_arr];
  int yi = cu_yi[i_arr];

  double V[3];
  double pt[3];
  pt[0] = cu_cloud[3*i_arr]; pt[1] = cu_cloud[3*i_arr + 1]; pt[2] = cu_cloud[3*i_arr + 2];
  cu_normalize(V, pt, 3);

  if (cu_normals != NULL && cu_dot(V, &cu_normals[3*i_arr], 3) >= -.1) {  // normals pointing away
    cu_vis_prob[i_arr] = 0.0;
    return;
  }


  if (xi == -1 && yi == -1) {
    cu_vis_prob[i_arr] = 0.0;
    return;
  }

  double model_range = cu_norm(pt, 3);
  double obs_range = range_image.ptr[xi * range_image.m + yi];

  if (search_radius > 0) {
    int x0 = MAX(xi - search_radius, 0);
    int x1 = MIN(xi + search_radius, ri_data.w - 1);
    int y0 = MAX(yi - search_radius, 0);
    int y1 = MIN(yi + search_radius, ri_data.h - 1);
    int x, y;
    for (x = x0; x <= x1; x++)
      for (y = y0; y <= y1; y++)
	obs_range = MAX(obs_range, range_image.ptr[x * range_image.m + y]);
  }

  double dR = model_range - obs_range;
  cu_vis_prob[i_arr] = (dR < 0 ? 1.0 : cu_normpdf(dR/vis_thresh, 0, 1) / .3989);  // .3989 = normpdf(0,0,1)
}

__global__ void cu_get_viewpoints(int *vi, int num_samples, double *samples_x, double *samples_q, cu_double_matrix_t range_edges_model_views) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= num_samples)
    return;

  double vp[3];
  
  cu_model_pose_to_viewpoint(vp, &samples_x[3*i], &samples_q[4*i]);
  double vi_max = -(1<<19);

  int j;
  for (j = 0; j < range_edges_model_views.n; ++j) {
    double tmp = cu_dot(&range_edges_model_views.ptr[j * range_edges_model_views.m], vp, 3);
    if (tmp > vi_max) {
      vi[i] = j;
      vi_max = tmp;
    }
  }
}

__global__ void cu_get_noise_models(scope_noise_model_t *noise_models, double *cloud, double *normals, int *idx, int *vi, cu_double_matrix_t ved, cu_double_arr_t normalvar, int num_samples, int n) {

  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= n || i >= num_samples)
    return;

  // prep for lookup edge distances for closest model viewpoint                                                                                                                                                    

  double surface_angles, edge_dists;
  // compute sigmas                 
                           
  int i_arr = i * n + j;
                                                                                                                                                    
  double normalized[3];
  cu_normalize(normalized, &cloud[3*i_arr], 3);
  surface_angles = 1 + cu_dot(normalized, &normals[3 * i_arr], 3);
  edge_dists = ved.ptr[idx[i_arr] * ved.m + vi[i]];
  noise_models[i_arr].range_sigma = .5*cu_sigmoid(surface_angles, b_SR) + .5*cu_sigmoid(edge_dists, b_ER);
  noise_models[i_arr].normal_sigma = .5*cu_sigmoid(surface_angles, b_SN) + .5*cu_sigmoid(edge_dists, b_EN);
  noise_models[i_arr].lab_sigma[0] = .5*cu_sigmoid(surface_angles, b_SL) + .5*cu_sigmoid(edge_dists, b_EL);
  noise_models[i_arr].lab_sigma[1] = .5*cu_sigmoid(surface_angles, b_SA) + .5*cu_sigmoid(edge_dists, b_EA);
  noise_models[i_arr].lab_sigma[2] = .5*cu_sigmoid(surface_angles, b_SB) + .5*cu_sigmoid(edge_dists, b_EB);
  
  noise_models[i_arr].normal_sigma = MAX(noise_models[i_arr].normal_sigma, normalvar.ptr[idx[i_arr]]);
}

__global__ void cu_transform_cloud(double *cloud2, double *cloud, double *x, double *q, int num_samples, int n, int *n_arr)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= n)
    return;
  if (n_arr && j >= n_arr[i])
    return;

  int i_arr = j + i * n;

  double R[3][3];
  cu_quaternion_to_rotation_matrix(R,&q[4*i]);
 
  double tmp[3];
  cu_matrix_vec_mult_3(tmp, R, &cloud[i_arr*3], 3);
  cloud2[3*i_arr] = tmp[0];
  cloud2[3*i_arr+1] = tmp[1];
  cloud2[3*i_arr+2] = tmp[2];
  if (x != NULL) {
    cu_add(&cloud2[i_arr*3], &cloud2[i_arr*3], &x[3*i], 3);
  }
}

__global__ void cu_compute_xyz_score_individual(double *xyz_score, double *cloud, int *xi, int *yi, double *vis_pmf, scope_noise_model_t *noise_models, int num_samples, int num_validation_points, 
						cu_double_matrix_t range_image, cu_range_image_data_t range_image_data, cu_int_matrix_t range_image_cnt, scope_params_t *params)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= num_validation_points || i >= num_samples)
    return;

  int xyz_score_window = params->xyz_score_window;
  
  int i_arr = j + i * num_validation_points;

  xyz_score[i_arr] = 0.0;

  if (vis_pmf[i_arr] > .01/(double)num_validation_points) {
    double range_sigma = params->range_sigma * noise_models[i_arr].range_sigma;
    double model_range = cu_norm(&cloud[3*i_arr], 3);
    double dmax = 2*range_sigma;
    double dmin = dmax;
    int x, y;
    int r = xyz_score_window;
    for (x = xi[i_arr] - r; x<=xi[i_arr] + r; ++x) {
      for (y = yi[i_arr] - r; y <= yi[i_arr] + r; ++y) {
	if (x >= 0 && x < (range_image_data.w) && y>=0 && y<(range_image_data.h) && range_image_cnt.ptr[x * range_image_cnt.m + y] > 0) {
	  double obs_range = range_image.ptr[x * range_image.m + y];
	  double d = fabs(model_range - obs_range);
	  if (d < dmin) 
	    dmin = d;	    
	}
      }
    }
    double d = dmin;
    xyz_score[i_arr] = vis_pmf[i_arr] * log(cu_normpdf(d, 0, range_sigma));
    
  }
}

__global__ void cu_compute_xyz_score_final(double *xyz_scores, int num_samples, double *b_xyz, scope_params_t *params, int score_round) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= num_samples)
    return;

  xyz_scores[i] -= log(cu_normpdf(0, 0, params->range_sigma));

  if ((score_round == 2 && params->score2_use_score_comp_models) || (score_round == 3 && params->score3_use_score_comp_models))
    xyz_scores[i] = cu_logistic(xyz_scores[i], b_xyz);

  double w = 0;
  if (score_round == 2)
    w = params->score2_xyz_weight;
  else
    w = params->score3_xyz_weight;

  xyz_scores[i] *= w;
}

__global__ void cu_compute_normal_score_individual(double *normal_score, double *wtot_individual, double *cloud_normals, double *vis_pmf, scope_noise_model_t *noise_models, int num_samples, 
						int num_validation_points, int *xi, int *yi, cu_int_matrix_t range_image_cnt, cu_double_matrix3d_t range_image_normals, scope_params_t *params, int score_round)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= num_validation_points || i >= num_samples)
    return;

  int i_arr = j + i * num_validation_points;

  //TODO: make this a param
  double normalvar_thresh = params->normalvar_thresh;

  normal_score[i_arr] = 0.0;
  wtot_individual[i_arr] = 0.0;

  if (vis_pmf[i_arr] > .01/ (double) num_validation_points && noise_models[i_arr].normal_sigma <= normalvar_thresh) {
    double normal_sigma = params->normal_sigma * noise_models[i_arr].normal_sigma;
    double dmax = 2*normal_sigma;
    double d = dmax;
    if ((xi[i_arr] != -1 && yi[i_arr] != -1) && range_image_cnt.ptr[xi[i_arr] * range_image_cnt.m + yi[i_arr]] > 0) {
      d = 1.0 - cu_dot(&cloud_normals[3*i_arr], &(range_image_normals.ptr[xi[i_arr] * range_image_normals.m * range_image_normals.p + yi[i_arr] * range_image_normals.p]), 3);
      d = MIN(d, dmax);
    }
    normal_score[i_arr] = vis_pmf[i_arr] * log(cu_normpdf(d, 0, normal_sigma));
    wtot_individual[i_arr] = vis_pmf[i_arr];
  }
}

__global__ void cu_compute_normal_score_final(double *normal_scores, double *wtot, int num_samples, double *b_normal, scope_params_t *params, int score_round) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= num_samples)
    return;
  
  if (wtot[i] > 0.0)
    normal_scores[i] /= wtot[i];
  normal_scores[i] -= log(cu_normpdf(0, 0, params->normal_sigma));

  if ((score_round == 2 && params->score2_use_score_comp_models) || (score_round == 3 && params->score3_use_score_comp_models))
    normal_scores[i] = cu_logistic(normal_scores[i], b_normal);

  double w = 0;
  if (score_round == 2)
    w = params->score2_normal_weight;
  else
    w = params->score3_normal_weight;

  normal_scores[i] *= w;
}

__global__ void cu_compute_vis_score(double *vis_score, double *vis_sums, int n, scope_params_t *params, int score_round)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= n)
    return;

  vis_score[i] = log(vis_sums[i] / (double) n);
  
  double w = 0;
  if (score_round == 2)
    w = params->score2_vis_weight;
  else
    w = params->score3_vis_weight;

  vis_score[i] *= w;
}

__global__ void cu_set_mask_for_segment_affinity(int *mask, int *segments, int *num_segments, int num_obs_segments, int num_samples) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= num_segments[i])
    return;

  // Assumes mask is initialized to all zeros before kernel execution  
  mask[segments[j + i * num_obs_segments] + i * num_obs_segments] = 1;
}

// compute the segment affinity score for a scope sample
__global__ void cu_compute_segment_affinity_score_per_seg(double *seg_affinity_score_per_seg, int *segments, int *num_segments, cu_double_matrix_t segment_affinities, int num_obs_segments, int *mask, 
							  int num_samples)	    
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= num_obs_segments)
    return;

  int k;
  
  seg_affinity_score_per_seg[j + i * num_obs_segments] = 0.0;
  if (mask[j + i * num_obs_segments] == 0) {
    for (k = 0; k < num_segments[i]; ++k) {
      int s = segments[k + i * num_obs_segments];
      double a = MIN(segment_affinities.ptr[s * segment_affinities.m + j], .9);
      if (a > 0.5)
	seg_affinity_score_per_seg[j + i * num_obs_segments] += log((1-a)/a);
    }
  }
}

__global__ void cu_compute_segment_affinity_score_final(double *seg_affinity_score, scope_params_t *params, int score_round, int num_samples) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= num_samples)
    return;
    
  seg_affinity_score[i] *= .05;

  double weight = 0;
  if (score_round == 2)
    weight = params->score2_segment_affinity_weight;
  else
    weight = params->score3_segment_affinity_weight;

  seg_affinity_score[i] *= weight;
}

__global__ void cu_generate_n_for_range_edge(int *n_out, int *vi, int num_samples, int num_validation_points, cu_int_arr_t range_edges_view_cnt) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;  
  
  if (i >= num_samples)
    return;
  
  int v_idx = vi[i];
  int num_edge_points = range_edges_view_cnt.ptr[v_idx];

  int n = num_validation_points;

  if (n >= num_edge_points || n == 0) {
    n = num_edge_points;
  }
  n_out[i] = n;
}

__global__ void cu_get_range_edge_rnd(int *idx, int *needed, int num_samples, int total_pts, int n, uint *rands, int *vi,cu_int_arr_t range_edges_view_idx)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= needed[i])
    return;

  // NOTE(sanja): This might need some fixing if I use the function in a broader sense, like on a CPU version
  if (needed[i] <= n) {  // use all the points
    idx[j + i * total_pts] = j;
  } else {
    idx[j + i * total_pts] = ((rands[i << 1] % needed[i]) + (j * (big_primes[rands[(i << 1) + 1] % 100] % needed[i]))) % needed[i];
  }
  int vp_idx = range_edges_view_idx.ptr[vi[i]];
  idx[j + i * total_pts] += vp_idx;

}

__global__ void cu_get_range_edge_points(double *P, int num_samples, int *n, int *idx, int n_edge, cu_double_matrix_t range_edges_points)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples)
    return;
  if (j >= n[i])
    return;

  // get the actual points in the correct pose
  P[3 * i * n_edge + 3 * j] = range_edges_points.ptr[3 * idx[j + i * n_edge]];
  P[3 * i * n_edge + 3 * j + 1] = range_edges_points.ptr[3 * idx[j + i * n_edge] + 1];
  P[3 * i * n_edge + 3 * j + 2] = range_edges_points.ptr[3 * idx[j + i * n_edge] + 2];
}

__global__ void cu_compute_edge_score_individual(double *edge_score, double *vis_pmf, int *xi, int *yi, cu_double_matrix_t edge_image, int num_samples, int *n, int n_edge) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= num_samples || j >= n[i])
    return;

  edge_score[j + i * n_edge] = 0.0;
  if (xi[j + i *n_edge] != -1 && yi[j + i * n_edge] != -1) {
    edge_score[j + i * n_edge] = vis_pmf[j + i * n_edge] * edge_image.ptr[xi[j + i *n_edge]*edge_image.m + yi[j + i *n_edge]];
  }
}

__global__ void cu_compute_edge_score_final(double *edge_score, double *vis_score, double *vis_prob_sums, double *occ_score, int num_samples, int *n_arr, double *b_edge, double *b_edge_occ, 
					    scope_params_t *params, int score_round) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= num_samples)
    return;

  if ((score_round == 2 && params->score2_use_score_comp_models) || (score_round == 3 && params->score3_use_score_comp_models)) {
    edge_score[i] = cu_logistic(edge_score[i], b_edge);
    if (occ_score)
      occ_score[i] = cu_logistic(occ_score[i], b_edge_occ);
  }

  vis_score[i] = log(vis_prob_sums[i] / (double) n_arr[i]);

  double w1=0.0, w2=0.0, w3=0.0;
  w1=1.0, w2=1.0, w3=1.0;
  if (score_round == 2) {
    w1 = params->score2_edge_weight;
    w2 = params->score2_edge_vis_weight;
    w3 = params->score2_edge_occ_weight;
  }
  else {
    w1 = params->score3_edge_weight;
    w2 = params->score3_edge_vis_weight;
    w3 = params->score3_edge_occ_weight;
  }

  if (occ_score)
    edge_score[i] = (w1 * edge_score[i]) + (w2 * vis_score[i]) + (w3 * occ_score[i]);
  else
    edge_score[i] = (w1 * edge_score[i]) + (w2 * vis_score[i]);    
}
  
__global__ void cu_score_round1(double *scores, int *xi, int *yi, double *cloud, cu_double_matrix_t range_image, int num_samples, int num_validation_points) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= num_samples)
    return;

  double dthresh = .05;  //TODO: make this a param
  double sample_score = 0;

  double dest[3];
  
  int i_arr = i*num_validation_points;
  int j;

  // TODO(sanja): optimize!
  for (j = 0; j < num_validation_points; ++j) {
    dest[0] = cloud[3*(i_arr + j)]; dest[1] = cloud[3*(i_arr + j)+1]; dest[2] = cloud[3*(i_arr + j) + 2];
    if ((xi[i_arr + j] != -1 && yi[i_arr + j] != -1) && 
	range_image.ptr[xi[i_arr + j]*range_image.m + yi[i_arr + j]] > dthresh + cu_norm(dest, 3))
      sample_score -= 1.0;
  }

  sample_score /= (double)num_validation_points;
  
  scores[i] = sample_score;
}
 
__global__ void cu_add_all_scores(double *cu_scores, double *cu_xyz_score, double *cu_normal_score, double *cu_vis_score, double *cu_seg_affinity_score, double *cu_edge_scores, int num_samples) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= num_samples)
    return;

  cu_scores[i] = cu_xyz_score[i] + cu_normal_score[i] + cu_vis_score[i] + cu_seg_affinity_score[i] + cu_edge_scores[i];
}

void score_samples(double *scores, scope_sample_t *samples, int num_samples, cu_model_data_t *cu_model, cu_obs_data_t *cu_obs, scope_params_t *cu_params, scope_params_t *params, int num_validation_points, 
		   int model_points, int num_obs_segments, int edge_scoring, int round) {
  // NEXT(sanja): Figure out why the hell seg mask makes things crappy

  hipError_t cudaerr;

  dim3 threads_per_block(256, 1, 1);
  dim3 block_size(ceil(1.0 * num_validation_points / threads_per_block.x), num_samples);

  dim3 thread_size_small(64);
  dim3 block_size_small(ceil(1.0 * num_samples/thread_size_small.x));

  dim3 thread_size_sum(256);
  dim3 block_size_sum(1, num_samples);
  dim3 thread_size_sum_small(64);
  
  int num_total = num_samples * num_validation_points;

  double **samples_x = new_matrix2(num_samples, 3);  
  double **samples_q = new_matrix2(num_samples, 4);
  int i;
  for (i = 0; i < num_samples; ++i) {
    memcpy(samples_x[i], samples[i].x, 3 * sizeof(double));
  }
  for (i = 0; i < num_samples; ++i) {
    memcpy(samples_q[i], samples[i].q, 4 * sizeof(double));
  }  

  double *cu_samples_x;
  cu_malloc(&cu_samples_x, num_samples * 3 * sizeof(double), "samples_x");
  hipMemcpy(cu_samples_x, samples_x[0], 3 * num_samples * sizeof(double), hipMemcpyHostToDevice);

  double *cu_samples_q;
  cu_malloc(&cu_samples_q, num_samples * 4 * sizeof(double), "samples_y");
  hipMemcpy(cu_samples_q, samples_q[0], 4 * num_samples * sizeof(double), hipMemcpyHostToDevice);
 
  uint *cu_rands;
  cu_malloc(&cu_rands, 2 * num_samples * sizeof(uint), "rands");
  if (model_points > num_validation_points) {
    hiprandGenerate(gen, cu_rands, 2*num_samples);
  }

  int *cu_idx;
  cu_malloc(&cu_idx, num_total * sizeof(int), "idxs");
  cu_get_validation_points<<<block_size, threads_per_block>>>(cu_idx, cu_model->num_points, num_validation_points, num_samples, cu_rands);
  cudaerr = hipGetLastError();
  if ( hipSuccess != cudaerr )
    printf( "Validation!\n" );
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

  // extract transformed model validation features
  double *cu_cloud;
  cu_malloc(&cu_cloud, 3 * num_total * sizeof(double), "cloud");
  cu_get_sub_cloud_at_pose<<<block_size, threads_per_block>>>(cu_cloud, cu_model->points, cu_samples_x, cu_samples_q, cu_idx, num_samples, num_validation_points);
  if ( hipSuccess != hipGetLastError() )
    printf( "Subcloud!\n" );

  int *cu_xi;
  cu_malloc(&cu_xi, num_total * sizeof(int), "xi");
  int *cu_yi;
  cu_malloc(&cu_yi, num_total * sizeof(int), "yi");
  cu_populate_xi_yi<<<block_size, threads_per_block>>>(cu_xi, cu_yi, cu_cloud, cu_obs->range_image_data, num_samples, num_validation_points, NULL);

  double *cu_scores;
  cu_malloc(&cu_scores, num_samples * sizeof(double), "scores");

  if (round == 1) {
    cu_score_round1<<<block_size_small, thread_size_small>>>(cu_scores, cu_xi, cu_yi, cu_cloud, cu_obs->range_image, num_samples, num_validation_points);
    if ( hipSuccess != hipGetLastError() )
      printf( "Round 1 score!\n" );
  } else {    
    double *cu_normals;
    cu_malloc(&cu_normals, 3 * num_total * sizeof(double), "normals");
    cu_get_sub_cloud_normals_rotated<<<block_size, threads_per_block>>>(cu_normals, cu_model->normals, cu_samples_q, cu_idx, num_samples, num_validation_points);
    if ( hipSuccess != hipGetLastError() )
      printf( "Normals!\n" );

    double *cu_vis_prob;
    cu_malloc(&cu_vis_prob, num_total * sizeof(double), "vis_prob");
    cu_compute_visibility_prob<<<block_size, threads_per_block>>>(cu_vis_prob, cu_cloud, cu_normals, cu_xi, cu_yi, cu_obs->range_image_data, cu_obs->range_image, params->vis_thresh, 0,
								  num_samples, num_validation_points, NULL);
    if ( hipSuccess != hipGetLastError() )
      printf( "vis_prob!\n" );
    double *cu_vis_prob_sums;
    cu_malloc(&cu_vis_prob_sums, num_samples * sizeof(double), "vis_prob_sums");
    // NOTE(sanja): If we ever get a newer graphics card, we can make this call from another kernel. We can also probably pull the whole normalize thing into a host function.
    cu_add_matrix_rows_medium<<<block_size_sum, thread_size_sum, thread_size_sum.x * sizeof(double)>>>(cu_vis_prob_sums, cu_vis_prob, num_samples, num_validation_points, NULL); 
    // TODO(sanja): Optimize. ArrayFire?
    if ( hipSuccess != hipGetLastError() )
      printf( "Vis prob sums!\n" );
    
    double *cu_vis_pmf;
    cu_malloc(&cu_vis_pmf, num_total * sizeof(double), "vis_pmf");
    cu_divide_matrix_with_vector<<<block_size, threads_per_block>>>(cu_vis_pmf, cu_vis_prob, cu_vis_prob_sums, num_samples, num_validation_points, NULL);
    if ( hipSuccess != hipGetLastError() )
      printf( "Vis pmf!\n" );    

    int *cu_vi;
    cu_malloc(&cu_vi, num_samples * sizeof(int), "vi");
    cu_get_viewpoints<<<block_size_small, thread_size_small>>>(cu_vi, num_samples, cu_samples_x, cu_samples_q, cu_model->range_edges_model_views);
    if ( hipSuccess != hipGetLastError() )
      printf( "Viewpoints!\n" );

    scope_noise_model_t *cu_noise_models;
    cu_malloc(&cu_noise_models, num_total * sizeof(scope_noise_model_t), "noise_models");
    cu_get_noise_models<<<block_size, threads_per_block>>>(cu_noise_models, cu_cloud, cu_normals, cu_idx, cu_vi, cu_model->ved, cu_model->normalvar, num_samples, 
							   num_validation_points);
    if ( hipSuccess != hipGetLastError() )
      printf( "Noise model!\n" );    
    // TODO(sanja): Save results before weights kick in
    double *cu_xyz_score_per_point;
    cu_malloc(&cu_xyz_score_per_point, num_total * sizeof(double), "xyz_scores_pp");
    double *cu_xyz_score;
    cu_malloc(&cu_xyz_score, num_samples * sizeof(double), "xyz_scores");
    cu_compute_xyz_score_individual<<<block_size, threads_per_block>>>(cu_xyz_score_per_point, cu_cloud, cu_xi, cu_yi, cu_vis_pmf, cu_noise_models, num_samples, num_validation_points, 
								       cu_obs->range_image, cu_obs->range_image_data, cu_obs->range_image_cnt, cu_params);
    if ( hipSuccess != hipGetLastError() )
      printf( "xyz individual!\n" );
    cu_add_matrix_rows_medium<<<block_size_sum, thread_size_sum, thread_size_sum.x * sizeof(double)>>>(cu_xyz_score, cu_xyz_score_per_point, num_samples, num_validation_points, NULL);
    if ( hipSuccess != hipGetLastError() )
      printf( "xyz sums!\n" );

    cu_compute_xyz_score_final<<<block_size_small, thread_size_small>>>(cu_xyz_score, num_samples, cu_model->score_comp_models->b_xyz, cu_params, round);
    if ( hipSuccess != hipGetLastError() )
      printf( "xyz final!\n" );
    
    double *cu_normal_score_per_point;
    cu_malloc(&cu_normal_score_per_point, num_total * sizeof(double), "normal_score_pp");

    double *cu_wtot_per_point;
    cu_malloc(&cu_wtot_per_point, num_total * sizeof(double), "wtot_pp");
    cu_compute_normal_score_individual<<<block_size, threads_per_block>>>(cu_normal_score_per_point, cu_wtot_per_point, cu_normals, cu_vis_pmf, cu_noise_models, num_samples, num_validation_points, cu_xi, cu_yi,
									  cu_obs->range_image_cnt, cu_obs->range_image_normals, cu_params, round);
    if ( hipSuccess != hipGetLastError() )
      printf( "normal individual!\n" );
    
    double *cu_normal_score, *cu_wtot;
    cu_malloc(&cu_normal_score, num_samples * sizeof(double), "normal_score");
    cu_malloc(&cu_wtot, num_samples * sizeof(double), "wtot");
    cu_add_matrix_rows_medium<<<block_size_sum, thread_size_sum, thread_size_sum.x * sizeof(double)>>>(cu_normal_score, cu_normal_score_per_point, num_samples, num_validation_points, NULL);
    if ( hipSuccess != hipGetLastError() )
      printf( "add 1!\n" );
    cu_add_matrix_rows_medium<<<block_size_sum, thread_size_sum, thread_size_sum.x * sizeof(double)>>>(cu_wtot, cu_wtot_per_point, num_samples, num_validation_points, NULL);
    if ( hipSuccess != hipGetLastError() )
      printf( "add 2!\n" );

    cu_compute_normal_score_final<<<block_size_small, thread_size_small>>>(cu_normal_score, cu_wtot, num_samples, cu_model->score_comp_models->b_normal, cu_params, round);

    double *cu_vis_score;
    cu_malloc(&cu_vis_score, num_samples * sizeof(double), "vis_score");
    cu_compute_vis_score<<<block_size_small, thread_size_small>>>(cu_vis_score, cu_vis_prob_sums, num_validation_points, cu_params, round);
    if ( hipSuccess != hipGetLastError() )
      printf( "vis score!\n" );

    // TODO(sanja): Figure out how to speed up the prep for segment calculation
    double *cu_seg_affinity_score_per_seg;
    cu_malloc(&cu_seg_affinity_score_per_seg, num_samples * num_obs_segments * sizeof(double), "seg_aff_per_seg");
    int *cu_mask;
    cu_malloc(&cu_mask, num_samples * num_obs_segments * sizeof(int), "mask");
    hipMemset(cu_mask, 0, num_samples * num_obs_segments * sizeof(int));
    int *num_segments;
    safe_calloc(num_segments, num_samples, int);
    for (i = 0; i < num_samples; ++i) {
      num_segments[i] = samples[i].num_segments;
    }
    int *cu_num_segments;
    cu_malloc(&cu_num_segments, num_samples * sizeof(int), "num_segments");
    hipMemcpy(cu_num_segments, num_segments, num_samples * sizeof(int), hipMemcpyHostToDevice);
    free(num_segments);
    int *tmp_segments_idx;
    safe_malloc(tmp_segments_idx, num_samples * num_obs_segments, int);
    memset(tmp_segments_idx, -1, num_samples * num_obs_segments * sizeof(int));
    for (i = 0; i < num_samples; ++i) {
      memcpy(&(tmp_segments_idx[i * num_obs_segments]), samples[i].segments_idx, samples[i].num_segments * sizeof(int));
    }
    int *cu_segments_idx;
    cu_malloc(&cu_segments_idx, num_samples * num_obs_segments * sizeof(int), "segments_idx");
    hipMemcpy(cu_segments_idx, tmp_segments_idx, num_samples * num_obs_segments * sizeof(int), hipMemcpyHostToDevice);
    if ( hipSuccess != hipGetLastError() )
      printf( "seg idx memcpy!\n" );

    free(tmp_segments_idx);
    double *cu_seg_affinity_score;
    cu_malloc(&cu_seg_affinity_score, num_samples * sizeof(double), "seg_aff_per_seg");
    
    dim3 block_size_seg(ceil(1.0 * num_obs_segments / thread_size_sum.x), num_samples);
    //cu_set_mask_for_segment_affinity<<<block_size_seg, thread_size_sum>>>(cu_mask, cu_segments_idx, cu_num_segments, num_obs_segments, num_samples);
    cu_set_mask_for_segment_affinity<<<block_size_seg, thread_size_sum>>>(cu_mask, cu_segments_idx, cu_num_segments, num_obs_segments, num_samples);
    if ( hipSuccess != hipGetLastError() )
      printf( "seg mask!\n" );

    cu_compute_segment_affinity_score_per_seg<<<block_size, thread_size_small>>>(cu_seg_affinity_score_per_seg, cu_segments_idx, cu_num_segments, cu_obs->segment_affinities, num_obs_segments, 
										 cu_mask, num_samples);
    if ( hipSuccess != hipGetLastError() )
      printf( "seg per seg!\n" );

    cu_add_matrix_rows_slow<<<block_size_small, thread_size_small>>>(cu_seg_affinity_score, cu_seg_affinity_score_per_seg, num_samples, num_obs_segments, NULL);
    cu_compute_segment_affinity_score_final<<<block_size_small, thread_size_small>>>(cu_seg_affinity_score, cu_params, round, num_samples);
    if ( hipSuccess != hipGetLastError() )
      printf( "seg affinity!\n" );

    double *cu_edge_scores;
    cu_malloc(&cu_edge_scores, num_samples * sizeof(double), "edge_scores");
    hipMemset(cu_edge_scores, 0, num_samples * sizeof(double));
    if ( hipSuccess != hipGetLastError() )
      printf( "memset!\n" );
    if (edge_scoring) {
      int n_edge = cu_model->max_num_edges;
      int *cu_n;
      cu_malloc(&cu_n, num_samples * sizeof(int), "n");
      cu_generate_n_for_range_edge<<<block_size_small, thread_size_small>>>(cu_n, cu_vi, num_samples, num_validation_points, cu_model->range_edges_view_cnt);
      if ( hipSuccess != hipGetLastError() )
	printf( "n!\n" );

      uint *cu_rands_edge;
      cu_malloc(&cu_rands_edge, 2 * num_samples * sizeof(uint), "rands");
      hiprandGenerate(gen, cu_rands_edge, 2*num_samples);
      int *cu_idx_edge;
      cu_malloc(&cu_idx_edge, num_samples * n_edge * sizeof(int), "idx_edge");
      dim3 block_size_n_edge(ceil(1.0 * n_edge / thread_size_sum.x), num_samples);
      cu_get_range_edge_rnd<<<block_size_n_edge, thread_size_sum>>>(cu_idx_edge, cu_n, num_samples, n_edge, num_validation_points, cu_rands_edge, cu_vi, cu_model->range_edges_view_idx);
      if ( hipSuccess != hipGetLastError() )
	printf( "idx edge!\n" );
      double *cu_P;
      cu_malloc(&cu_P, num_samples * n_edge * 3*sizeof(double), "cu_P");
      cu_get_range_edge_points<<<block_size_n_edge, thread_size_sum>>>(cu_P, num_samples, cu_n, cu_idx_edge, n_edge, cu_model->range_edges_points);
      if ( hipSuccess != hipGetLastError() )
	printf( "edge pts\n" );
      cu_transform_cloud<<<block_size_n_edge, thread_size_sum>>>(cu_P, cu_P, cu_samples_x, cu_samples_q, num_samples, n_edge, cu_n);
      if ( hipSuccess != hipGetLastError() )
	printf( "transform cloud\n" );
      double *cu_edge_score_individual;
      cu_malloc(&cu_edge_score_individual, num_samples * n_edge * sizeof(double), "edge_score");
      double *cu_vis_prob_edge, *cu_vis_prob_sums_edge, *cu_vis_pmf_edge;
      cu_malloc(&cu_vis_prob_edge, num_samples * n_edge * sizeof(double), "vis_prob_edge");
      cu_malloc(&cu_vis_prob_sums_edge, num_samples * sizeof(double), "vis_prob_sums_edge");
      cu_malloc(&cu_vis_pmf_edge, num_samples * n_edge * sizeof(double), "vis_pmf_edge");
      int *cu_xi_edge;
      cu_malloc(&cu_xi_edge, num_samples * n_edge * sizeof(int), "xi");
      int *cu_yi_edge;
      cu_malloc(&cu_yi_edge, num_samples * n_edge * sizeof(int), "yi");
      cu_populate_xi_yi<<<block_size_n_edge, thread_size_sum>>>(cu_xi_edge, cu_yi_edge, cu_P, cu_obs->range_image_data, num_samples, n_edge, cu_n);
      if ( hipSuccess != hipGetLastError() )
	printf( "edge xi yi!\n" );
      
      int vis_pixel_radius = 2;
      cu_compute_visibility_prob<<<block_size_n_edge, thread_size_sum>>>(cu_vis_prob_edge, cu_P, NULL, cu_xi_edge, cu_yi_edge, cu_obs->range_image_data, cu_obs->range_image, params->vis_thresh, 
									   vis_pixel_radius, num_samples, n_edge, cu_n);
      if ( hipSuccess != hipGetLastError() )
	printf( "edge score vis prob!\n" );


      cu_add_matrix_rows_slow<<<block_size_small, thread_size_small>>>(cu_vis_prob_sums_edge, cu_vis_prob_edge, num_samples, n_edge, cu_n);
      cu_divide_matrix_with_vector<<<block_size_n_edge, thread_size_sum>>>(cu_vis_pmf_edge, cu_vis_prob_edge, cu_vis_prob_sums_edge, num_samples, n_edge, cu_n);
      cu_compute_edge_score_individual<<<block_size_sum, thread_size_sum>>>(cu_edge_score_individual, cu_vis_pmf_edge, cu_xi_edge, cu_yi_edge, cu_obs->edge_image, num_samples, cu_n, n_edge);
      if ( hipSuccess != hipGetLastError() )
	printf( "edge score individual!\n" );
      cu_add_matrix_rows_slow<<<block_size_small, thread_size_small>>>(cu_edge_scores, cu_edge_score_individual, num_samples, n_edge, cu_n);

      double *cu_vis_scores;
      cu_malloc(&cu_vis_scores, num_samples * sizeof(double), "vis_scores");
      cu_compute_edge_score_final<<<block_size_small, thread_size_small>>>(cu_edge_scores, cu_vis_scores, cu_vis_prob_sums_edge, NULL, num_samples, cu_n, cu_model->score_comp_models->b_edge, 
									    cu_model->score_comp_models->b_edge_occ, cu_params, round);
      if ( hipSuccess != hipGetLastError() )
	printf( "edge score final!\n" );

	      
      cu_free(cu_n, "n");
      cu_free(cu_idx_edge, "idx_edge");
      cu_free(cu_rands_edge, "rands_edge");
      cu_free(cu_P, "P");
      cu_free(cu_edge_score_individual, "edge_score_individual");
      cu_free(cu_vis_prob_sums_edge, "vis_prob_sums_edge");
      cu_free(cu_vis_pmf_edge, "vis_pmf_edge");
      cu_free(cu_vis_prob_edge, "vis_prob_edge");
      cu_free(cu_xi_edge, "xi_edge");
      cu_free(cu_yi_edge, "yi_edge");
      cu_free(cu_vis_scores, "vis_scores");
    }
    
    cu_add_all_scores<<<block_size_small, thread_size_small>>>(cu_scores, cu_xyz_score, cu_normal_score, cu_vis_score, cu_seg_affinity_score, cu_edge_scores, num_samples);
      
    if ( hipSuccess != hipGetLastError() )
      printf( "Final addition!\n" );

    // NEXT(sanja): Make calls for each score component async.

    cu_free(cu_normals, "normals");
    cu_free(cu_vis_prob, "vis_prob");
    cu_free(cu_vis_prob_sums, "vis_prob_sums");
    cu_free(cu_vis_pmf, "vis_pmf");
    cu_free(cu_vi, "vi");
    cu_free(cu_noise_models, "noise_models");

    cu_free(cu_xyz_score_per_point, "xyz_scores_pp");
    cu_free(cu_xyz_score, "xyz_scores");
    cu_free(cu_normal_score_per_point, "normal_scores_pp");
    cu_free(cu_normal_score, "normal_scores");
    cu_free(cu_wtot_per_point, "wtot_pp");
    cu_free(cu_wtot, "wtot");
    cu_free(cu_vis_score, "vis_score");
    cu_free(cu_seg_affinity_score_per_seg, "seg_aff_per_seg");
    cu_free(cu_seg_affinity_score, "seg_aff");
    cu_free(cu_mask, "mask");
    cu_free(cu_num_segments, "num_segments");
    cu_free(cu_segments_idx, "segments_idx");
    cu_free(cu_edge_scores, "edge_scores");
  }

  hipMemcpy(scores, cu_scores, num_samples * sizeof(double), hipMemcpyDeviceToHost);
  
  cu_free(cu_samples_x, "samples_x"); cu_free(cu_samples_q, "samples_y");
  cu_free(cu_rands, "rands free");
  cu_free(cu_idx, "idx");
  cu_free(cu_cloud, "cloud");
  cu_free(cu_xi, "xi"); cu_free(cu_yi, "yi");
  cu_free(cu_scores, "scores");
  hipDeviceSynchronize();
}
