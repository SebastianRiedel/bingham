#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "include/bingham/cuda_wrapper.h"
//#include "hip/hip_runtime_api.h"
#include "bingham/olf.h"
#include "hiprand.h"

#include <math.h>

//#define CUDA_LAUNCH_BLOCKING 1

#define MAX(x,y) ((x) > (y) ? (x) : (y))
#define MIN(x,y) ((x) < (y) ? (x) : (y))

#define cu_free(x, msg) do{ if (hipFree(x) != hipSuccess) printf(msg); } while (0)

__device__ int first = 1;
__device__ __constant__ int big_primes[100] = {996311, 163573, 481123, 187219, 963323, 103769, 786979, 826363, 874891, 168991, 442501, 318679, 810377, 471073, 914519, 251059, 321983, 220009, 211877, 875339, 605603, 578483, 219619, 860089, 644911, 398819, 544927, 444043, 161717, 301447, 201329, 252731, 301463, 458207, 140053, 906713, 946487, 524389, 522857, 387151, 904283, 415213, 191047, 791543, 433337, 302989, 445853, 178859, 208499, 943589, 957331, 601291, 148439, 296801, 400657, 829637, 112337, 134707, 240047, 669667, 746287, 668243, 488329, 575611, 350219, 758449, 257053, 704287, 252283, 414539, 647771, 791201, 166031, 931313, 787021, 520529, 474667, 484361, 358907, 540271, 542251, 825829, 804709, 664843, 423347, 820367, 562577, 398347, 940349, 880603, 578267, 644783, 611833, 273001, 354329, 506101, 292837, 851017, 262103, 288989};

__device__ __constant__ double b_SR[3] = {0.2878,    -5.6214,      7.7247};
__device__ __constant__ double b_SN[3] = {0.1521,    -7.1290,     10.7090};
__device__ __constant__ double b_SL[3] = {0.2238,    -5.1827,      6.8242};
__device__ __constant__ double b_SA[3] = {0.1618,    -6.3992,      8.0207};
__device__ __constant__ double b_SB[3] = {0.2313,    -6.3463,      8.0651};

__device__ __constant__ double b_ER[3] = {0.3036,     0.2607,   -125.8843};
__device__ __constant__ double b_EN[3] = {0.1246,     1.4406,   -185.8350};
__device__ __constant__ double b_EL[3] = {0.2461,     0.2624,   -140.0192};
__device__ __constant__ double b_EA[3] = {0.1494,     0.2114,   -139.4324};
__device__ __constant__ double b_EB[3] = {0.2165,     0.2600,   -135.5203};


// util.h stuff **********************************
/*__device__ static void init_rand() {
  if (first) {
    first = 0;
    srand (time(NULL));
  }
  } */

hiprandGenerator_t gen;

__device__ void cu_randperm(int *x, int n, int d, uint r1, uint r2) {
  int i;
  if (d > n) {
    printf("Error: d > n in randperm()\n");
    return;
  }
  
  // sample a random starting point
  int i0 = r1 % n;

  // use a random prime step to cycle through x

  int step = big_primes[r2 % 100];

  int idx = i0;
  for (i = 0; i < d; i++) {
    x[i] = idx;
    idx = (idx + step) % n;
  }
}

// computes the max of x
int arr_max_i(int *x, int n)
{
  int i;

  int y = x[0];
  for (i = 1; i < n; i++)
    if (x[i] > y)
      y = x[i];

  return y;
}

// create a new n-by-m 2d matrix of doubles
__device__ double **cu_new_matrix2(int n, int m)
{
  if (n*m == 0) return NULL;
  int i;
  double *raw, **X;
  raw = (double *) malloc(n*m*sizeof(double));
  memset(raw, 0, n*m*sizeof(double));
  X = (double **) malloc(n * sizeof(double*));
  for (i = 0; i < n; i++)
    X[i] = raw + m*i;
  return X;
}

// free a 2d matrix of doubles
__device__ void cu_free_matrix2(double **X)
{
  if (X == NULL) return;
  free(X[0]);
  free(X);
}

// computes the dot product of z and y
__device__ double cu_dot(double x[], double y[], int n) {
  int i;
  double z = 0.0;
  for (i = 0; i < n; i++)
    z += x[i]*y[i];
  return z;
}

__device__ void cu_quaternion_to_rotation_matrix(double R[][3], double q[]) {
  double a = q[0];
  double b = q[1];
  double c = q[2];
  double d = q[3];

  R[0][0] = a*a + b*b - c*c - d*d;
  R[0][1] = 2*b*c - 2*a*d;
  R[0][2] = 2*b*d + 2*a*c;
  R[1][0] = 2*b*c + 2*a*d;
  R[1][1] = a*a - b*b + c*c - d*d;
  R[1][2] = 2*c*d - 2*a*b;
  R[2][0] = 2*b*d - 2*a*c;
  R[2][1] = 2*c*d + 2*a*b;
  R[2][2] = a*a - b*b - c*c + d*d;
}

__device__ void cu_matrix_vec_mult_3(double *y, double A[][3], double *x, int n, int m) {
  int i;
  /*if (y == x) {
    double *z = (double *) malloc(m * sizeof(double));
    memcpy(z, x, m*sizeof(double));
    for (i = 0; i < n; i++)
      y[i] = cu_dot(A[i], z, m);
    free(z);
  }
  else*/
  if (y == x) {
    printf("**************FIX CU_MATRIX_VEC_MULT CALL!\n");
  }
  for (i = 0; i < n; i++)
    y[i] = cu_dot(A[i], x, m);
}

__device__ void cu_matrix_vec_mult_flat(double *y, double *A, double *x, int n, int m) {
  int i;
  /*if (y == x) {
    double *z = (double *) malloc(m * sizeof(double));
    memcpy(z, x, m*sizeof(double));
    for (i = 0; i < n; i++)
      y[i] = cu_dot(A[i], z, m);
    free(z);
  }
  else*/
  if (y == x) {
    printf("**************FIX CU_MATRIX_VEC_MULT CALL!\n");
  }
  for (i = 0; i < n; i++)
    y[i] = cu_dot(&A[i * m], x, m);
}

// adds two vectors, z = x+y
__device__ void cu_add(double z[], double x[], double y[], int n) {
  int i;
  for (i = 0; i < n; i++)
    z[i] = x[i] + y[i];
}

// subtracts two vectors, z = x-y
__device__ void cu_sub(double z[], double x[], double y[], int n)
{
  int i;
  for (i = 0; i < n; i++)
    z[i] = x[i] - y[i];
}

// computes the sum of x's elements
__device__ double cu_sum(double x[], int n)
{
  int i;
  double y = 0;
  for (i = 0; i < n; i++)
    y += x[i];
  return y;
}

__device__ double cu_norm(double x[], int n) {
  double d = 0.0;
  int i;

  for (i = 0; i < n; i++)
    d += x[i]*x[i];

  return sqrt(d);
}

// compute the pdf of a normal random variable
__device__ double cu_normpdf(double x, double mu, double sigma)
{
  double dx = x - mu;

  return exp(-dx*dx / (2*sigma*sigma)) / (sqrt(2*M_PI) * sigma);
}

// sets y = x/sum(x)
__device__ void cu_normalize_pmf(double y[], double x[], int n)
{
  double d = cu_sum(x, n);
  int i;
  for (i = 0; i < n; i++)
    y[i] = x[i]/d;
}

// multiplies a vector by a scalar, y = c*x
__device__ void cu_mult(double y[], double x[], double c, int n)
{
  int i;
  for (i = 0; i < n; i++)
    y[i] = c*x[i];
}

// sets y = x/norm(x)
__device__ void cu_normalize(double y[], double x[], int n)
{
  double d = cu_norm(x, n);
  int i;
  for (i = 0; i < n; i++)
    y[i] = x[i]/d;
}

// invert a quaternion
__device__ void cu_quaternion_inverse(double q_inv[4], double q[4])
{
  q_inv[0] = q[0];
  q_inv[1] = -q[1];
  q_inv[2] = -q[2];
  q_inv[3] = -q[3];
}

/*
 * get the plane equation coefficients (c[0]*x + c[1]*y + c[2]*z + c[3] = 0) from (point,normal)
 */
__device__ void cu_xyzn_to_plane(double *c, double *point, double *normal)
{
  c[0] = normal[0];
  c[1] = normal[1];
  c[2] = normal[2];
  c[3] = -cu_dot(point, normal, 3);
}

// reorder the rows of X, Y = X(idx,:)
__device__ void cu_reorder_rows(double *Y, double *X, int *idx, int n, int m)
{
  int i;
  if (X == Y) {
    printf("********** fix the call to cu_reorder_rows!\n");
  }
  for (i = 0; i < n; i++)
    memcpy(&Y[m*i], &X[m*idx[i]], m*sizeof(double));
}

__device__ double* get_row (cu_double_matrix_t *matrix, int i) {
  //return (double *)(((char *) matrix->ptr) + i * matrix->pitch);
  return &(matrix->ptr[i * matrix->m]);
}

__device__ double get_element(cu_double_matrix_t *matrix, int i, int j) {
  //double *row = (double *) (((char *) matrix->ptr) + i * matrix->pitch);
  //return *(row + j);
  return matrix->ptr[i * matrix->m + j];
}

/*
 * compute viewpoint (in model coordinates) for model placement (x,q) assuming observed viewpoint = (0,0,0)
 */
__device__ void cu_model_pose_to_viewpoint(double *vp, double x[], double q[])
{
  double q_inv[4];
  cu_quaternion_inverse(q_inv, q);
  double R_inv[3][3];
  cu_quaternion_to_rotation_matrix(R_inv,q_inv);
  cu_matrix_vec_mult_3(vp, R_inv, x, 3, 3);
  cu_mult(vp, vp, -1, 3);
}

__device__ void get_validation_points(int *idx, int num_points, int num_validation_points, uint *r)
{
  int i;
  if (num_validation_points == num_points)  // use all the points
    for (i = 0; i < num_validation_points; i++)
      idx[i] = i;
  else
    cu_randperm(idx, num_points, num_validation_points, r[0], r[1]);
}

__device__ void get_sub_cloud_at_pose(cu_double_matrix_t *points, double *cloud, int *idx, int n, double x[], double q[])
{
  double R[3][3];
  cu_quaternion_to_rotation_matrix(R,q);
  int i;
  double *row, *dest;
  for (i = 0; i < n; i++) {
    row = get_row(points, idx[i]);
    dest = &cloud[3*i];
    memcpy(dest, row, 3*sizeof(double));
    double tmp[3];
    cu_matrix_vec_mult_3(tmp, R, dest, 3, 3);
    cu_add(dest, tmp, x, 3);
  }
}

__device__ void get_sub_cloud_normals_rotated(cu_double_matrix_t *normals, double *cloud_normals, int *idx, int n, double q[])
{
  double R[3][3];
  cu_quaternion_to_rotation_matrix(R,q);
  int i;
  double *row, *dest;
  for (i = 0; i < n; i++) {
    row = get_row(normals, idx[i]);
    dest = &cloud_normals[3*i];
    double tmp[3];
    memcpy(tmp, row, 3*sizeof(double));
    cu_matrix_vec_mult_3(dest, R, tmp, 3, 3);
  }
}

__device__ void get_sub_cloud_lab(cu_double_matrix_t *lab, double *cloud_lab, int *idx, int n)
{
  cu_reorder_rows(cloud_lab, lab->ptr, idx, n, 3);
}

__device__ void range_image_xyz2sub(int *i, int *j, cu_range_image_data_t *range_image, double *xyz)
{
  //TODO: use range image viewpoint

  double d = cu_norm(xyz, 3);
  double x = atan2(xyz[0], xyz[2]);
  double y = acos(xyz[1] / d);

  int cx = (int)floor((x - range_image->min0) / range_image->res);
  int cy = (int)floor((y - range_image->min1) / range_image->res);

  *i = cx;
  *j = cy;

  if (!(cx>=0 && cy>=0 && (cx < range_image->w) && (cy < range_image->h))) {
    *i = -1;
    *j = -1;
  }
}

__device__ double compute_xyz_score(double *cloud, int *xi, int *yi, double *vis_pmf, scope_noise_model_t *noise_models, int num_validation_points, 
				    cu_double_matrix_t *range_image, cu_range_image_data_t *range_image_data, cu_int_matrix_t *range_image_cnt, scope_params_t *params, int score_round)
{
  double score = 0.0;
  //double range_sigma = params->range_sigma;
  //double dmax = 2*range_sigma;
  int i;
  for (i = 0; i < num_validation_points; i++) {
    if (vis_pmf[i] > .01/(double)num_validation_points) {
      double range_sigma = params->range_sigma * noise_models[i].range_sigma;
      double model_range = cu_norm(&cloud[3*i], 3);
      double dmax = 2*range_sigma;
      double dmin = dmax;
      int x, y;
      for (x = xi[i] - 1; x<=xi[i] + 1; ++x) {
	for (y = yi[i] - 1; y <= yi[i] + 1; ++y) {
	  if (x >= 0 && x < (range_image_data->w) && y>=0 && y<(range_image_data->h) && range_image_cnt->ptr[x * range_image_cnt->m + y] > 0) {
	    double obs_range = range_image->ptr[x * range_image->m + y];
	    double d = fabs(model_range - obs_range);
	    if (d < dmin) 
	      dmin = d;	    
	  }
	}
      }
      double d = dmin;
      score += vis_pmf[i] * log(cu_normpdf(d, 0, range_sigma));

    }
  }
  score -= log(cu_normpdf(0, 0, params->range_sigma));

  double w = 0;
  if (score_round == 2)
    w = params->score2_xyz_weight;
  else
    w = params->score3_xyz_weight;

  return w * score;
}

__device__ double compute_normal_score(double *cloud_normals, double *vis_pmf, scope_noise_model_t *noise_models, int num_validation_points, int *xi, int *yi,
				       cu_int_matrix_t *range_image_cnt, cu_double_matrix3d_t *range_image_normals, scope_params_t *params, int score_round)
{
  //TODO: make this a param
  double normalvar_thresh = .3;

  double score = 0.0;
  //double normal_sigma = params->normal_sigma;
  //double dmax = 2*normal_sigma;  // TODO: make this a param
  int i;
  double wtot = 0.0;
  for (i = 0; i < num_validation_points; i++) {
    if (vis_pmf[i] > .01/ (double) num_validation_points && noise_models[i].normal_sigma <= normalvar_thresh) {
      double normal_sigma = params->normal_sigma * noise_models[i].normal_sigma;
      double dmax = 2*normal_sigma;
      double d = dmax;
      if ((xi[i] != -1 && yi[i] != -1) && range_image_cnt->ptr[xi[i] * range_image_cnt->m + yi[i]] > 0) {
	// get distance from model normal to range image cell normal
	d = 1.0 - cu_dot(&cloud_normals[3*i], &(range_image_normals->ptr[xi[i] * range_image_normals->m * range_image_normals->p + yi[i] * range_image_normals->p]), 3);
	//d /= noise_models[i].normal_sigma;
	d = MIN(d, dmax);
      }
      score += vis_pmf[i] * log(cu_normpdf(d, 0, normal_sigma));
      wtot += vis_pmf[i];
    }
  }
  score /= wtot;
  score -= log(cu_normpdf(0, 0, params->normal_sigma));

  double w = 0;
  if (score_round == 2)
    w = params->score2_normal_weight;
  else
    w = params->score3_normal_weight;

  return w * score;
}

__device__ double compute_lab_score(int *xi, int *yi, double *lab, double *vis_pmf, scope_noise_model_t *noise_models, int n, cu_int_matrix_t *range_image_idx, cu_double_matrix_t *pcd_obs_bg_lab, 
				    scope_params_t *params, int score_round) 
{
  double scores[3] = {0, 0, 0};
  int i, j;
  //double L_weight = params->L_weight;
  //double lab_sigma = params->lab_sigma;
  //double dmax = 2*lab_sigma; // * sqrt(2.0 + L_weight*L_weight);  // TODO: make this a param
  for (i = 0; i < n; i++) {
    if (vis_pmf[i] > .01/(double)n) {
      //double d = dmax;
      double dlab[3], dmax[3], lab_sigma[3];
      for (j = 0; j < 3; j++) {
	lab_sigma[j] = params->lab_sigma * noise_models[i].lab_sigma[j];
	dmax[j] = 2*lab_sigma[j];
	dlab[j] = dmax[j];
      }
      int obs_idx = range_image_idx->ptr[xi[i] * range_image_idx->m + yi[i]];
      if (obs_idx >= 0) {
	double *obs_lab = &pcd_obs_bg_lab->ptr[obs_idx * pcd_obs_bg_lab->m];
	cu_sub(dlab, &lab[3*i], obs_lab, 3);
	//dlab[0] = L_weight * (lab[i][0] - obs_lab[0]) / noise_models[i].lab_sigma[0];
	//dlab[1] = (lab[i][1] - obs_lab[1]) / noise_models[i].lab_sigma[1];
	//dlab[2] = (lab[i][2] - obs_lab[2]) / noise_models[i].lab_sigma[2];
	//dlab[2] = 0;
	//d = norm(dlab, 3);
	//d = MIN(d, dmax);
	for (j = 0; j < 3; j++)	  
	  dlab[j] = MIN(dlab[j], dmax[j]);

	//dbug
	//if (params->verbose && (i%100==0)) {
	//  printf("model lab[%d] = [%.2f, %.2f, %.2f], obs_lab = [%.2f, %.2f, %.2f]\n", i, lab[i][0], lab[i][1], lab[i][2], obs_lab[0], obs_lab[1], obs_lab[2]);
	//}
      }
      //score += vis_pmf[i] * log(normpdf(d, 0, lab_sigma));
      for (j = 0; j < 3; j++)
	scores[j] += vis_pmf[i] * log(cu_normpdf(dlab[j], 0, lab_sigma[j]));

    }
  }
  //score -= log(normpdf(0, 0, params->lab_sigma));

  double lab_weights2[3] = {params->score2_L_weight, params->score2_A_weight, params->score2_B_weight};
  double lab_weights3[3] = {params->score3_L_weight, params->score3_A_weight, params->score3_B_weight};

  double *w = NULL;
  if (score_round == 2)
    w = lab_weights2;
  else
    w = lab_weights3;

  return cu_dot(scores, w, 3);
}

__device__ double compute_vis_score(double *vis_prob, int n, scope_params_t *params, int score_round)
{
  double score = log(cu_sum(vis_prob, n) / (double) n);

  double w = 0;
  if (score_round == 2)
    w = params->score2_vis_weight;
  else
    w = params->score3_vis_weight;

  return w * score;
}

/*void labdist_color_shift(double *shift, pcd_color_model_t *color_model, int *idx, int n, double **obs_lab, double *obs_weights, double pmin, scope_params_t *params)
{
  //TODO: make these params
  double lambda = 1.0;
  double shift_threshold = 0.1;

  double **C_inv = new_matrix2(3,3);
  double **B = new_matrix2(3,3);
  inv(B, color_model->avg_cov, 3);
  double **A = new_matrix2(3,3);
  double z[3];  // m-bar
  double w;

  memset(shift, 0, 3*sizeof(double));

  int i, j, iter, max_iter = 10;
  for (iter = 0; iter < max_iter; iter++) {

    // reset shift statistics
    memset(A[0], 0, 9*sizeof(double));
    memset(z, 0, 3*sizeof(double));
    w = 0;

    for (i = 0; i < n; i++) {

      if (obs_weights[i] == 0.0)
	continue;

      int cnt1 = color_model->cnts[0][idx[i]];
      int cnt2 = color_model->cnts[1][idx[i]];
      if (cnt1 < 4)
	cnt1 = 0;
      if (cnt2 < 4)
	cnt2 = 0;
      if (cnt1 == 0 && cnt2 == 0)
	continue;

      double *m1 = color_model->means[0][idx[i]];
      double *m2 = color_model->means[1][idx[i]];
      double **C1 = color_model->covs[0][idx[i]];
      double **C2 = color_model->covs[1][idx[i]];

      // assign observed color to a cluster
      double y[3];  // current obs_lab[i]
      add(y, obs_lab[i], shift, 3);
      double p1 = (cnt1 > 0 ? mvnpdf(y, m1, C1, 3) : 0);
      double p2 = (cnt2 > 0 ? mvnpdf(y, m2, C2, 3) : 0);
      
      // check if assigned cluster could be a specularity cluster (i.e., has higher L-value)
      if ((p1 > p2 && p2 > 0 && m1[0] > m2[0]) || (p2 > p1 && p1 > 0 && m2[0] > m1[0]))
	continue;

      double *m = (p1 > p2 ? m1 : m2);
      double **C = (p1 > p2 ? C1 : C2);

      double maxp = mvnpdf(m, m, C, 3);
      double p = mvnpdf(y, m, C, 3);

      // check if point is an outlier of the cluster
      if (p < pmin*maxp)
	continue;

      // add observed color and color model covariance matrix to the shift statistics
      for (j = 0; j < 3; j++)
	z[j] = z[j] + obs_weights[i]*(m[j] - obs_lab[i][j]);
      inv(C_inv, C, 3);
      for (j = 0; j < 9; j++)
	A[0][j] = A[0][j] + obs_weights[i]*C_inv[0][j];
      w += obs_weights[i];
    }

    mult(z, z, 1/w, 3);  // avg. z
    mult(A[0], A[0], lambda/w, 9);  // avg. A and multiply by lambda

    // solve for best shift = inv(lambda*A+B)*lambda*A*z
    double new_shift[3];
    matrix_vec_mult(z, A, z, 3, 3);
    add(A[0], A[0], B[0], 9);
    inv(C_inv, A, 3);
    matrix_vec_mult(new_shift, C_inv, z, 3, 3);
    double d2 = dist2(shift, new_shift, 3);
    memcpy(shift, new_shift, 3*sizeof(double));

    //printf("shift = [%f, %f, %f]\n", shift[0], shift[1], shift[2]);  //dbug

    if (d2 < shift_threshold*shift_threshold)
      break;
  }

  // apply shift to obs_lab
  for (i = 0; i < n; i++)
    if (obs_weights[i] > 0.0)
      add(obs_lab[i], obs_lab[i], shift, 3);

  free_matrix2(A);
  free_matrix2(B);
  free_matrix2(C_inv);
}

double compute_labdist_score(double **cloud, pcd_color_model_t *color_model, int *idx, double *vis_pmf, scope_noise_model_t *noise_models, int n,
			     range_image_t *obs_range_image, pcd_t *pcd_obs, scope_params_t *params, int score_round)
{
  //TODO: make this a param
  double pmin = .1;

  // get obs colors
  double **obs_lab = new_matrix2(n,3);
  double obs_weights[n];
  memset(obs_weights, 0, n*sizeof(double));
  int i;
  for (i = 0; i < n; i++) {
    if (vis_pmf[i] > .01/(double)n) {
      int xi,yi;
      range_image_xyz2sub(&xi, &yi, obs_range_image, cloud[i]);

      int obs_idx = obs_range_image->idx[xi][yi];
      if (obs_idx >= 0) {
	memcpy(obs_lab[i], pcd_obs->lab[obs_idx], 3*sizeof(double));
	obs_weights[i] = vis_pmf[i];
      }
    }
  }

  // get color shift (and apply it to obs_lab)
  double color_shift[3];
  labdist_color_shift(color_shift, color_model, idx, n, obs_lab, obs_weights, pmin, params);

  if (params->verbose) {
    memset(mps_labdist_p_ratios_, 0, n*sizeof(double));
  }

  double zero[3] = {0,0,0};
  double score = 0.0;
  for (i = 0; i < n; i++) {
    if (vis_pmf[i] > .01/(double)n) {
      double logp = labdist_likelihood(color_model, idx[i], (obs_weights[i] > 0 ? obs_lab[i] : zero), pmin, params);
      score += vis_pmf[i] * logp;
    }
  }

  double w = 0;
  if (score_round == 2)
    w = params->score2_labdist_weight;
  else
    w = params->score3_labdist_weight;

  free_matrix2(obs_lab);

  return w * score;
}*/

__device__ double compute_visibility_prob(double *point, double *normal, int xi, int yi, cu_range_image_data_t *ri_data, cu_double_matrix_t *range_image, double vis_thresh, int search_radius)
//double compute_visibility_prob(double *point, double *normal, range_image_t *obs_range_image, double vis_thresh, int search_radius)
{
  double V[3];
  cu_normalize(V, point, 3);

  if (normal != NULL && cu_dot(V, normal, 3) >= -.1)  // normals pointing away
    return 0.0;

  if (xi == -1 && yi == -1)
    return 0.0;

  double model_range = cu_norm(point, 3);
  double obs_range = range_image->ptr[xi * range_image->m + yi];

  if (search_radius > 0) {
    int x0 = MAX(xi - search_radius, 0);
    int x1 = MIN(xi + search_radius, ri_data->w - 1);
    int y0 = MAX(yi - search_radius, 0);
    int y1 = MIN(yi + search_radius, ri_data->h - 1);
    int x, y;
    for (x = x0; x <= x1; x++)
      for (y = y0; y <= y1; y++)
	obs_range = MAX(obs_range, range_image->ptr[x * range_image->m + y]);
  }

  double dR = model_range - obs_range;
  return (dR < 0 ? 1.0 : cu_normpdf(dR/vis_thresh, 0, 1) / .3989);  // .3989 = normpdf(0,0,1)
}

__device__ inline double cu_sigmoid(double x, const double *b)
{
  return b[0] + (1 - b[0]) / (1 + exp(-b[1]-b[2]*x));
}

__device__ void get_noise_models(scope_noise_model_t *noise_models, double *cloud, double *cloud_normals, double x[], double q[], int *idx, int n, 
				 cu_double_matrix_t *ved, cu_double_matrix_t *range_edges_model_views, cu_double_arr_t *normalvar)
{
  int i;

  // prep for lookup edge distances for closest model viewpoint
  double vp[3];

  cu_model_pose_to_viewpoint(vp, x, q);
  int vi;
  double vi_max = -(1<<29);
  // Did this without functions to avoid stuff like a[n] that's not supported in C++
  for (i = 0; i < range_edges_model_views->n; ++i) {
    double tmp = cu_dot(&range_edges_model_views->ptr[i * range_edges_model_views->m], vp, 3);
    if (tmp > vi_max) {
      vi = i;
      vi_max = tmp;
    }
  }

  double surface_angles, edge_dists;
  // compute sigmas
  for (i = 0; i < n; i++) {
    double normalized[3];
    cu_normalize(normalized, &cloud[3*i], 3);
    surface_angles = 1 + cu_dot(normalized, &cloud_normals[3*i], 3);
    edge_dists = ved->ptr[idx[i] * ved->m + vi];
    noise_models[i].range_sigma = .5*cu_sigmoid(surface_angles, b_SR) + .5*cu_sigmoid(edge_dists, b_ER);
    noise_models[i].normal_sigma = .5*cu_sigmoid(surface_angles, b_SN) + .5*cu_sigmoid(edge_dists, b_EN);
    noise_models[i].lab_sigma[0] = .5*cu_sigmoid(surface_angles, b_SL) + .5*cu_sigmoid(edge_dists, b_EL);
    noise_models[i].lab_sigma[1] = .5*cu_sigmoid(surface_angles, b_SA) + .5*cu_sigmoid(edge_dists, b_EA);
    noise_models[i].lab_sigma[2] = .5*cu_sigmoid(surface_angles, b_SB) + .5*cu_sigmoid(edge_dists, b_EB);

    noise_models[i].normal_sigma = MAX(noise_models[i].normal_sigma, normalvar->ptr[idx[i]]);
  }
  
}

__device__ void cu_transform_cloud(double *cloud2, double *cloud, int n, double x[], double q[])
{
  double R[3][3];
  cu_quaternion_to_rotation_matrix(R,q);
  int i;
  for (i = 0; i < n; i++) {
    double tmp[3];
    cu_matrix_vec_mult_3(tmp, R, &cloud[i*3], 3, 3);
    memcpy(&cloud2[3*i], tmp, 3*sizeof(double));
    if (x != NULL) {
      cu_add(&cloud2[i*3], &cloud2[i*3], x, 3);
    }
  }
}

__device__ void get_range_edge_points(double *P, int *idx, int *n_ptr, double x[], double q[], 
				      cu_double_matrix_t *range_edges_model_views, cu_int_arr_t *range_edges_view_cnt, cu_int_arr_t *range_edges_view_idx, cu_double_matrix_t *range_edges_points, uint *r)
{
  // compute viewpoint for model placement (x,q) assuming observed viewpoint = (0,0,0)
  double vp[3];
  cu_model_pose_to_viewpoint(vp, x, q);

  int i;
  double i_max = -(1<<29);
  // Did this without functions to avoid stuff like a[n] that's not supported in C++
  int ii;
  for (ii = 0; ii < range_edges_model_views->n; ++ii) {
    double tmp = cu_dot(&range_edges_model_views->ptr[ii * range_edges_model_views->m], vp, 3);
    if (tmp > i_max) {
      i = ii;
      i_max = tmp;
    }
  }
  
  int vp_idx = range_edges_view_idx->ptr[i];
  int num_edge_points = range_edges_view_cnt->ptr[i];

  //printf("vp = [%f, %f, %f], closest stored vp = [%f, %f, %f]\n", vp[0], vp[1], vp[2],
  //	 range_edges_model->views[i][0], range_edges_model->views[i][1], range_edges_model->views[i][2]);  //dbug

  // sample edge points to validate
  //int idx[num_edge_points];
  // TODO(sanja): This is a performance hit, figure out a way around it
  int n = *n_ptr;
  if (n >= num_edge_points || n == 0) {
    n = num_edge_points;
    for (i = 0; i < n; i++)
      idx[i] = i;
  }
  else
    cu_randperm(idx, num_edge_points, n, r[0], r[1]);

  // make idx be pcd point indices
  for (i = 0; i < n; i++)
    idx[i] += vp_idx;

  //printf("n = %d, idx[0] = %d, idx[n-1] = %d\n", n, idx[0], idx[n-1]); //dbug

  // get the actual points in the correct pose
  // TODO(sanja): performance hit...
  cu_reorder_rows(P, range_edges_points->ptr, idx, n, 3);

  *n_ptr = n;
}

__device__ void cu_get_sub_matrix(double *Y, double *X, int x0, int y0, int x1, int y1)
{
  int h = y1-y0+1;

  int x;
  for (x = x0; x <= x1; x++)
    memcpy(&Y[(x-x0) * h], &X[x * h + y0], h*sizeof(double));
}

__device__ void cu_dilate_matrix(double *Y, double *X, int n, int m, int n2, int m2)
{
  int i, j;
  for (i = 0; i < n; i++) {
    for (j = 0; j < m; j++) {
      if (X[i * m2 + j] > 0.0)
	Y[i * m + j] = X[i * m2 + j];
      else {
	int cnt = 0;
	double p = 0.0;
	if (i > 0 && X[(i-1) * m2 + j] > 0.0) {  p += X[(i-1)*m2 + j]; cnt++;  }
	if (i < n-1 && X[(i+1) * m2 + j] > 0.0) {  p += X[(i+1) * m2 + j]; cnt++;  }
	if (j > 0 && X[i*m2 + j-1] > 0.0) {  p += X[i*m2 + j-1]; cnt++;  }
	if (j < m-1 && X[i*m2 + j+1] > 0.0) {  p += X[i*m2 + j+1]; cnt++;  }
	if (cnt > 0)
	  Y[i * m + j] = p / (double)cnt;
      }
    }
  }
}

__device__ void compute_occ_edges(int *occ_edges, double *V, double *V2, int *num_occ_edges, int *xi, int *yi, double *vis_prob, int n, cu_range_image_data_t *ri_data, scope_params_t *params)
{
  // create vis_prob image, V
  int i;
  int w = ri_data->w;
  int h = ri_data->h;
  int x0 = w, y0 = h, x1 = 0, y1 = 0;  // bounding box for model points in vis_prob_image
  for (i = 0; i < n; i++) {
    if ((xi[i] != -1 && yi[i] != -1) && vis_prob[i] > V[xi[i] * h + yi[i]]) {
      V[xi[i] * h + yi[i]] = vis_prob[i];
      if (xi[i] < x0)
	x0 = xi[i];
      if (xi[i] > x1)
	x1 = xi[i];
      if (yi[i] < y0)
	y0 = yi[i];
      if (yi[i] > y1)
	y1 = yi[i];
    }
  }

  // downsample vis_prob sub matrix (loses a row or column if w2 or h2 is odd)
  cu_get_sub_matrix(V2, V, x0, y0, x1, y1);
  int w2 = (x1-x0+1)/2;
  int h2 = (y1-y0+1)/2;
  int w22 = x1-x0+1;
  int h22 = y1-y0+1;

  int x,y;
  for (x = 0; x < w2; x++) {
    for (y = 0; y < h2; y++) {
      double v2 = MAX(V2[2*x * h22 + 2*y], V2[(2*x+1) * h22 + 2*y]);
      v2 = MAX(v2, V2[2*x * h22 + (2*y+1)]);
      V2[x*w22 + y] = MAX(v2, V2[(2*x+1)*h22 + (2*y+1)]);
    }
  }

  // dilate vis_prob sub matrix
  cu_dilate_matrix(V, V2, w2, h2, w22, h22);
  //dilate_matrix(V2, V, w2, h2);
  //dilate_matrix(V, V2, w2, h2);
  //dilate_matrix(V2, V, w2, h2);
  //dilate_matrix(V, V2, w2, h2);

  // compute edges where vis_prob crosses .5 threshold
  int cnt=0;
  for (x = 0; x < w2-1; x++) {
    for (y = 0; y < h2-1; y++) {
      if (V[x*h + y] >= .5) {
	if ((x > 0 && V[(x-1) * h + y] > 0.0 && V[(x-1)*h + y] < .5) || (x < w2-1 && V[(x+1) * h + y] > 0.0 && V[(x+1) * h + y] < .5) ||
	    (y > 0 && V[x * h + y-1] > 0.0 && V[x * h + y-1] < .5) || (y < h2-1 && V[x * h + y+1] > 0.0 && V[x*h + y+1] < .5)) {
	  ++cnt;
	}
      }
    }
  }
  *num_occ_edges = cnt;

  if (cnt==0)
    return;

  cnt = 0;
  for (x = 0; x < w2-1; x++) {
    for (y = 0; y < h2-1; y++) {
      if (V[x*h + y] >= .5) {
	if ((x > 0 && V[(x-1) * h + y] > 0.0 && V[(x-1) *h + y] < .5) || (x < w2-1 && V[(x+1) * h + y] > 0.0 && V[(x+1) * h + y] < .5) ||
	    (y > 0 && V[x * h + y-1] > 0.0 && V[x * h + y-1] < .5) || (y < h2-1 && V[x * h + y+1] > 0.0 && V[x*h + y+1] < .5)) {
	  occ_edges[2*cnt] = x0 + 2*x;
	  occ_edges[2*cnt + 1] = y0 + 2*y;
	  ++cnt;
	}
      }
    }
  }
}

__device__ double compute_edge_score(double *P, double *vis_prob, double *vis_pmf, int n, int *occ_edges, int num_occ_edges, cu_range_image_data_t *range_image_data,
				     cu_double_matrix_t *range_image, cu_double_matrix_t *edge_image, scope_params_t *params, int score_round)
{
  if (n == 0)
    return 0.0;

  // compute visibility of sampled model edges
  int vis_pixel_radius = 2;
  int i;
  for (i = 0; i < n; i++) {
    int x, y;
    range_image_xyz2sub(&x, &y, range_image_data, &P[3*i]);
    if (x == -1 && y == -1) {
      vis_prob[i] = 0.0;
      continue;
    }
    vis_prob[i] = compute_visibility_prob(&P[3*i], NULL, x, y, range_image_data, range_image, params->vis_thresh, vis_pixel_radius);
  }
  cu_normalize_pmf(vis_pmf, vis_prob, n);

  // compute obs_edge_image score for sampled model edges
  double score = 0;
  int xi,yi;
  for (i = 0; i < n; i++) {
    range_image_xyz2sub(&xi, &yi, range_image_data, &P[3*i]);
    if (xi != -1 && yi != -1) {
      score += vis_pmf[i] * edge_image->ptr[xi*edge_image->m + yi];
    }
  }
  double vis_score = log(cu_sum(vis_prob, n) / (double) n);

  //printf("gpu %d %lf\n", n, vis_score);
    
  // add occlusion edges to score
  double occ_score = 0.0;
  if (num_occ_edges > 0) {
    for (i = 0; i < num_occ_edges; i++) {
      int x = occ_edges[2 * i];
      int y = occ_edges[2 * i + 1];
      occ_score += edge_image->ptr[x * edge_image->m + y];
    }
    occ_score /= (double) num_occ_edges;
    occ_score = num_occ_edges*occ_score / (double)(n + num_occ_edges);
    score = n*score / (double)(n + num_occ_edges);
  }

  double w1=0, w2=0, w3=0;
  if (score_round == 2) {
    w1 = params->score2_edge_weight;
    w2 = params->score2_edge_vis_weight;
    w3 = params->score2_edge_occ_weight;
  }
  else {
    w1 = params->score3_edge_weight;
    w2 = params->score3_edge_vis_weight;
    w3 = params->score3_edge_occ_weight;
  }

  return (w1 * score) + (w2 * vis_score) + (w3 * occ_score);
}

__device__ double cu_model_placement_score(double x[], double q[], cu_model_data_t *cu_model, cu_obs_data_t *cu_obs, scope_params_t *cu_params, int score_round, 
					   int *xi, int *yi, int *idx, double *cloud, double *cloud_normals, double *cloud_lab, int num_validation_points, double *vis_prob, double *vis_pmf, uint *r,
					   scope_noise_model_t *noise_models,
					   int *idx_edge, double *P, double *V_edge, double *V2_edge, int *occ_edges, double *vis_prob_edge, double *vis_pmf_edge) {
  //int dbg_timed = 1;
  //double t0 = get_time_ms();  //dbug
  
  // get model validation points
  
  int i;
  get_validation_points(idx, cu_model->num_points, num_validation_points, r);
  
  /*if (dbg_timed) {
    printf("break 0, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/
  
  /*if (dbg_timed) {
    printf("break 1, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/
  
  // extract transformed model validation features
  get_sub_cloud_at_pose(&(cu_model->points), cloud, idx, num_validation_points, x, q);
  double *dest;
  for (i = 0; i < num_validation_points; ++i) {
    dest = &cloud[3*i];
    range_image_xyz2sub(&xi[i], &yi[i], &(cu_obs->range_image_data), dest);
  }
  
  if (score_round == 1) {  // after c=1, just use free space to score
    double dthresh = .05;  //TODO: make this a param
    double score = 0;

    for (i = 0; i < num_validation_points; i++) {
      dest = &cloud[3*i];
      if ((xi[i] != -1 && yi[i] != -1) && get_element(&(cu_obs->range_image), xi[i], yi[i]) > dthresh + cu_norm(dest, 3))
	score -= 1.0;
    }
    score /= (double)num_validation_points;
    return score;
  }
    
  get_sub_cloud_normals_rotated(&(cu_model->normals), cloud_normals, idx, num_validation_points, q);
  
  //double **cloud_sdw = get_sub_cloud_sdw(model_data->pcd_model, idx, num_validation_points, params);
  get_sub_cloud_lab(&(cu_model->lab), cloud_lab, idx, num_validation_points);
  //double **cloud_labdist = get_sub_cloud_labdist(model_data->pcd_model, idx, num_validation_points);
  //double **cloud_xyzn = get_xyzn_features(cloud, cloud_normals, num_validation_points, params);

  /*if (dbg_timed) {
    printf("break 2, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/

  // compute p(visibile)
  for (i = 0; i < num_validation_points; i++) {
    vis_prob[i] = compute_visibility_prob(&cloud[3 * i], &cloud_normals[3 * i], xi[i], yi[i], &(cu_obs->range_image_data), &(cu_obs->range_image), cu_params->vis_thresh, 0);
  }
  cu_normalize_pmf(vis_pmf, vis_prob, num_validation_points);

  //if (params->verbose)
  //  memcpy(mps_vis_prob_, vis_prob, num_validation_points*sizeof(double));
  
  /*if (dbg_timed) {
    printf("break 3, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/

  // compute noise models
  get_noise_models(noise_models, cloud, cloud_normals, x, q, idx, num_validation_points, &(cu_model->ved), &(cu_model->range_edges_model_views), &(cu_model->normalvar));

  /*
  if (dbg_timed) {
    printf("break 4, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/
  
  // compute nearest neighbors
  //int nn_idx[num_validation_points];  memset(nn_idx, 0, num_validation_points*sizeof(int));
  //double nn_d2[num_validation_points];  memset(nn_d2, 0, num_validation_points*sizeof(double));
  //int search_radius = 0;  // pixels
  //for (i = 0; i < num_validation_points; i++)
  //if (vis_prob[i] > .01)
  //range_image_find_nn(&nn_idx[i], &nn_d2[i], &cloud[i], &cloud_xyzn[i], 1, 6, obs_xyzn, obs_range_image, search_radius);
  //range_image_find_nn(&nn_idx[i], &nn_d2[i], &cloud[i], &cloud[i], 1, 3, pcd_obs->points, obs_range_image, search_radius);
  
  double normal_score = compute_normal_score(cloud_normals, vis_pmf, noise_models, num_validation_points, xi, yi, &(cu_obs->range_image_cnt), &(cu_obs->range_image_normals), cu_params, score_round);
  double xyz_score = compute_xyz_score(cloud, xi, yi, vis_pmf, noise_models, num_validation_points, &(cu_obs->range_image), &(cu_obs->range_image_data), &(cu_obs->range_image_cnt), 
				       cu_params, score_round);
  double lab_score = compute_lab_score(xi, yi, cloud_lab, vis_pmf, noise_models, num_validation_points, &(cu_obs->range_image_idx), &(cu_obs->range_image_pcd_obs_bg_lab), cu_params, score_round);
  //double labdist_score = compute_labdist_score(cloud, cloud_labdist, vis_pmf, noise_models, num_validation_points, obs_data->obs_range_image, obs_data->pcd_obs_bg, params, score_round);
  double vis_score = compute_vis_score(vis_prob, num_validation_points, cu_params, score_round);

  /* ----- Sanja's comment --------
  double labdist_score = 0;
  if (round > 2)
    labdist_score = compute_labdist_score(cloud, model_data->color_model, idx, vis_pmf, noise_models, num_validation_points, obs_data->obs_range_image, obs_data->pcd_obs_bg, params, score_round);
  
  // get fpfh score (TODO: add fpfh features to occ_model)
  int fpfh_num_validation_points = (params->num_validation_points > 0 ? params->num_validation_points : model_data->fpfh_model->num_points);
  int fpfh_idx[fpfh_num_validation_points];
  get_validation_points(fpfh_idx, model_data->fpfh_model, fpfh_num_validation_points);
  double **fpfh_cloud = get_sub_cloud_at_pose(model_data->fpfh_model, fpfh_idx, fpfh_num_validation_points, x, q);
  double **fpfh_cloud_normals = get_sub_cloud_normals_rotated(model_data->fpfh_model, fpfh_idx, fpfh_num_validation_points, q);
  double **fpfh_cloud_f = get_sub_cloud_fpfh(model_data->fpfh_model, fpfh_idx, fpfh_num_validation_points);
  double fpfh_vis_prob[fpfh_num_validation_points];
  for (i = 0; i < fpfh_num_validation_points; i++)
    fpfh_vis_prob[i] = compute_visibility_prob(fpfh_cloud[i], fpfh_cloud_normals[i], obs_data->obs_range_image, params->vis_thresh, 0);
  double fpfh_vis_pmf[fpfh_num_validation_points];
  normalize_pmf(fpfh_vis_pmf, fpfh_vis_prob, fpfh_num_validation_points);
  double fpfh_score = compute_fpfh_score(fpfh_cloud, fpfh_cloud_f, fpfh_vis_pmf, fpfh_num_validation_points, obs_data->obs_fg_range_image, obs_data->pcd_obs, params, score_round);
  
  double fpfh_score = 0;
  //double xyzn_score = compute_xyzn_score(nn_d2, vis_pmf, num_validation_points, params);
  //double xyz_score = compute_xyz_score(cloud, nn_idx, vis_pmf, num_validation_points, pcd_obs, params);
  //double sdw_score = compute_sdw_score(cloud_sdw, nn_idx, vis_pmf, num_validation_points, pcd_obs, params);
  //double lab_score = compute_lab_score(cloud_lab, nn_idx, vis_pmf, num_validation_points, pcd_obs, params);

  if (dbg_timed) {
    printf("break 5, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
  }
  */ // End Sanja's comment
  
  //TODO: move this to compute_edge_score()
  double edge_score = 0.0;
  if (cu_obs->edge_image.ptr) {
    int n = cu_params->num_validation_points;
    get_range_edge_points(P, idx_edge, &n, x, q, &(cu_model->range_edges_model_views), &(cu_model->range_edges_view_cnt), &(cu_model->range_edges_view_idx), &(cu_model->range_edges_points), &r[2]);
    cu_transform_cloud(P, P, n, x, q);
    /*
    if (cu_params->num_validation_points == 0) {
      int num_occ_edges;
      compute_occ_edges(occ_edges, V_edge, V2_edge, &num_occ_edges, xi, yi, vis_prob, num_validation_points, &(cu_obs->range_image_data), cu_params);
      edge_score = compute_edge_score(P, vis_prob_edge, vis_pmf_edge, n, occ_edges, num_occ_edges, &(cu_obs->range_image_data), &(cu_obs->range_image), &(cu_obs->edge_image), cu_params, score_round);
    }
    else*/
    edge_score = compute_edge_score(P, vis_prob_edge, vis_pmf_edge, n, NULL, 0, &(cu_obs->range_image_data), &(cu_obs->range_image), &(cu_obs->edge_image), cu_params, score_round);
  }

  /*if (dbg_timed) {
    printf("break 6, %.2f ms\n", get_time_ms() - t0);  //dbug
    t0 = get_time_ms();
    }*/

  /*double segment_score = 0;
  if (score_round >= 3)
    segment_score = compute_segment_score(x, q, cloud, model_data->model_xyz_index, &model_data->model_xyz_params, vis_prob,
					  num_validation_points, obs_data->obs_range_image, obs_data->obs_edge_image, params, score_round);
  */

  // (Sanja) double score = xyz_score + normal_score + edge_score + lab_score + vis_score + segment_score + fpfh_score + labdist_score;
  double score = xyz_score + normal_score + lab_score + vis_score + edge_score;

  //dbug
  //if (sample->c_type[0] == C_TYPE_SIFT)
  //  score += 100;

  /*if (dbg_timed) {
    printf("break 7, %.2f ms\n", get_time_ms() - t0);  //dbug
    } */
    
  return score;
}

__global__ void score_samples(double *cu_scores, cu_double_matrix_t cu_samples_x, cu_double_matrix_t cu_samples_q, int num_samples, cu_model_data_t cu_model, cu_obs_data_t cu_obs, scope_params_t cu_params, 
			      int score_round, int *cu_xi, int *cu_yi, int *cu_idx, double *cu_cloud, double *cu_cloud_normals, double *cu_cloud_lab, double *cu_vis_prob, double *cu_vis_pmf, uint *cu_rands, 
			      scope_noise_model_t *cu_noise_models,
			      int *cu_idx_edge, double *cu_P, double *cu_V_edge, double *cu_V2_edge, int *cu_occ_edge, double *cu_vis_prob_edge, double *cu_vis_pmf_edge) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_samples) {
    //printf("*****************gpu sample %d\n", i);
    int n_edge = cu_model.max_num_edges;
    double x[3], q[4];
    int j;
    double *row = get_row(&cu_samples_x, i);
    for (j = 0; j < 3; ++j)
      x[j] = row[j];
    row = get_row(&cu_samples_q, i);
    for (j = 0; j < 4; ++j)
      q[j] = row[j];
    //printf("%d\n", cu_params.num_validation_points);
    //printf("%d\n", cu_model.num_points);
    int num_validation_points = (cu_params.num_validation_points > 0 ? cu_params.num_validation_points : cu_model.num_points);
    int *xi, *yi, *idx;
    double *cloud, *cloud_normals, *cloud_lab;
    double *vis_pmf, *vis_prob;
    scope_noise_model_t *noise_models;
    xi = &cu_xi[i*num_validation_points];
    yi = &cu_yi[i*num_validation_points];
    idx = &cu_idx[i*num_validation_points];
    cloud = &cu_cloud[3*i*num_validation_points];
    cloud_normals = &cu_cloud_normals[3*i*num_validation_points];
    cloud_lab = &cu_cloud_lab[3*i*num_validation_points];
    vis_prob = &cu_vis_prob[i*num_validation_points];
    vis_pmf = &cu_vis_pmf[i*num_validation_points];
    noise_models = &cu_noise_models[i*num_validation_points];

    int w, h;
    w = cu_obs.range_image_data.w;
    h = cu_obs.range_image_data.h;

    int *idx_edge, *occ_edge; 
    double *P, *V_edge, *V2_edge, *vis_prob_edge, *vis_pmf_edge;
    idx_edge = &cu_idx_edge[i*n_edge];
    P = &cu_P[i*n_edge*3];
    V_edge = &cu_V_edge[i*w*h];
    V2_edge = &cu_V2_edge[i*w*h];
    occ_edge = &cu_occ_edge[i*w*h*2];
    vis_prob_edge = &cu_vis_prob_edge[i*n_edge];
    vis_pmf_edge = &cu_vis_pmf_edge[i*n_edge];

    cu_scores[i] = cu_model_placement_score(x, q, &cu_model, &cu_obs, &cu_params, score_round, xi, yi, idx, cloud, cloud_normals, cloud_lab, num_validation_points, 
					    vis_prob, vis_pmf, &cu_rands[4*i], noise_models,
					    idx_edge, P, V_edge, V2_edge, occ_edge, vis_prob_edge, vis_pmf_edge);

    //printf("%lf\n", cu_scores[i]);
  }
}

void copy_double_matrix_to_gpu(cu_double_matrix_t *dev_dest, double **host_src, int n, int m) {
  dev_dest->n = n;
  dev_dest->m = m;
  //hipMallocPitch(&(dev_dest->ptr), &(dev_dest->pitch), m * sizeof(double), n);
  //hipMemcpy2D(dev_dest->ptr, dev_dest->pitch, host_src[0], m * sizeof(double), m * sizeof(double), n, hipMemcpyHostToDevice); 
  if (hipMalloc(&(dev_dest->ptr), m*n*sizeof(double)) != hipSuccess) {
    printf("double 2d malloc\n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0], n * m * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
    printf("double 2d copy\n");
  }      
} 

void copy_int_matrix_to_gpu(cu_int_matrix_t *dev_dest, int **host_src, int n, int m) {
  dev_dest->n = n;
  dev_dest->m = m;
  //hipMallocPitch(&(dev_dest->ptr), &(dev_dest->pitch), m * sizeof(int), n);
  //hipMemcpy2D(dev_dest->ptr, dev_dest->pitch, host_src[0], m * sizeof(int), m * sizeof(int), n, hipMemcpyHostToDevice); 
  if (hipMalloc(&(dev_dest->ptr), m*n*sizeof(int)) != hipSuccess) {
    printf("int 2d malloc \n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0], n * m * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
    printf("int 2d copy\n");
  }      
}

void copy_double_matrix3d_to_gpu(cu_double_matrix3d_t *dev_dest, double ***host_src, int n, int m, int p) {
  dev_dest->n = n; dev_dest->m = m; dev_dest->p = p;
  /*  dest->extent = make_hipExtent(m * sizeof(double), n, p);
  hipMalloc3D(&(dest->ptr), dest->extent);
  hipPitchedPtr src_ptr;
  src_ptr.ptr = host_src[0][0];
  src_ptr.pitch = m * sizeof(double);
  src_ptr.xsize = m;
  src_ptr.ysize = n;
  hipMemcpy3DParms copy_params = {0};
  copy_params.srcPtr = src_ptr;
  copy_params.dstPtr = dest->ptr;
  copy_params.extent = dest->extent;
  copy_params.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&copy_params);*/
  if (hipMalloc(&(dev_dest->ptr), n * m * p * sizeof(double)) != hipSuccess) {
    printf("3d malloc\n");
  }      
  if (hipMemcpy(dev_dest->ptr, host_src[0][0], n * m * p * sizeof(double), hipMemcpyHostToDevice)) {
    printf("3d copy\n");
  }
}

void copy_double_arr_to_gpu(cu_double_arr_t *dev_dest, double *host_src, int n) {
  dev_dest->n = n;
  if (hipMalloc(&(dev_dest->ptr), n * sizeof(double)) != hipSuccess) {
    printf("double arr malloc\n");
  }
  if (hipMemcpy(dev_dest->ptr, host_src, n * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
    printf("double arr copy\n");
  }
}

void copy_int_arr_to_gpu(cu_int_arr_t *dev_dest, int *host_src, int n) {
  dev_dest->n = n;
  if (hipMalloc(&(dev_dest->ptr), n * sizeof(int)) != hipSuccess) {
    printf("int arr malloc\n");
  }
  if (hipMemcpy(dev_dest->ptr, host_src, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
    printf("int arr copy\n");
  }
}

void cu_score_samples(double *scores, scope_sample_t *samples, int num_samples, cu_model_data_t *cu_model, cu_obs_data_t *cu_obs, scope_params_t *cu_params, int score_round, int num_validation_points) {
  double t0 = get_time_ms();  
  int *cu_xi, *cu_yi, *cu_idx;
  double *cu_cloud, *cu_normals, *cu_lab;
  double *cu_vis_prob, *cu_vis_pmf;
  scope_noise_model_t *cu_noise_models;
  
  // NOTE(sanja): possible perf optimization: do all these in one giant Malloc. Downside: it might be hard to find this big chunk of memory. Solution: chunk it up a little bit.
  if (hipMalloc(&cu_xi, num_samples * num_validation_points * sizeof(int)) != hipSuccess) {
    printf("xi malloc\n");
  }      
  if (hipMalloc(&cu_yi, num_samples * num_validation_points * sizeof(int)) != hipSuccess) {
    printf("yi\n");
  }      
  if (hipMalloc(&cu_idx, num_samples * num_validation_points * sizeof(int)) != hipSuccess) {
    printf("idx\n");
  }      
  if (hipMalloc(&cu_cloud, num_samples * num_validation_points * 3 * sizeof(double)) != hipSuccess) {
    printf("cloud\n");
  }
  if (hipMalloc(&cu_lab, num_samples * num_validation_points * 3 * sizeof(double)) != hipSuccess) {
    printf("lab\n");
  }      
  if (hipMalloc(&cu_normals, num_samples * num_validation_points * 3 * sizeof(double)) != hipSuccess) {
    printf("normals\n");
  }      
  if (hipMalloc(&cu_vis_prob, num_samples * num_validation_points * sizeof(double)) != hipSuccess) {
    printf("vis_prob\n");
  }      
  if (hipMalloc(&cu_vis_pmf, num_samples * num_validation_points * sizeof(double)) != hipSuccess) {
    printf("vis_pmf\n");
  }      
  if (hipMalloc(&cu_noise_models, num_samples * num_validation_points * sizeof(scope_noise_model_t)) != hipSuccess) {
    printf("noise_models\n");
  }      
  
  // edge stuff
  int *cu_idx_edge, *cu_occ_edges;
  double *cu_P, *cu_V_edge, *cu_V2_edge, *cu_vis_prob_edge, *cu_vis_pmf_edge;
  int n_edge = cu_model->max_num_edges;
  if (hipMalloc(&cu_idx_edge, num_samples * n_edge * sizeof(int)) != hipSuccess) {
    printf("idx_edge\n");
  }
  if (hipMalloc(&cu_P, num_samples * n_edge * 3 * sizeof(double)) != hipSuccess) {
    printf("P\n");
  }
  int w, h;
  w = cu_obs->range_image_data.w;
  h = cu_obs->range_image_data.h;
  if (hipMalloc(&cu_V_edge, num_samples * w * h * sizeof(double)) != hipSuccess) {
    printf("V_edge\n");
  }
  if (hipMalloc(&cu_V2_edge, num_samples * w * h * sizeof(double)) != hipSuccess) {
    printf("V2_edge\n");
  }
  if (hipMalloc(&cu_occ_edges, num_samples * w * h * 2 * sizeof(int)) != hipSuccess) {
    printf("occ_edges\n");
  }
  if (hipMalloc(&cu_vis_prob_edge, num_samples * n_edge * sizeof(double)) != hipSuccess) {
    printf("vis_prob_edge\n");
  }
  if (hipMalloc(&cu_vis_pmf_edge, num_samples * n_edge * sizeof(double)) != hipSuccess) {
    printf("vis_pmf_edge\n");
  }

  double *cu_scores;
  if (hipMalloc(&cu_scores, num_samples * sizeof(double)) != hipSuccess) {
    printf("scores\n");
  }      

  uint *cu_rands;
  if (hipMalloc(&cu_rands, 2 * num_samples * sizeof(double)) != hipSuccess) {
    printf("rands\n");
  }      
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  hiprandGenerate(gen, cu_rands, 4 * num_samples);

  double **samples_x = new_matrix2(num_samples, 3);  
  double **samples_q = new_matrix2(num_samples, 4);
  int i;
  for (i = 0; i < num_samples; ++i) {
    memcpy(samples_x[i], samples[i].x, 3 * sizeof(double));
  }
  // I believe it is cache friendlier to copy like this
  for (i = 0; i < num_samples; ++i) {
    memcpy(samples_q[i], samples[i].q, 4 * sizeof(double));
  }  

  cu_double_matrix_t cu_samples_x;
  copy_double_matrix_to_gpu(&cu_samples_x, samples_x, num_samples, 3);
  cu_double_matrix_t cu_samples_q;
  copy_double_matrix_to_gpu(&cu_samples_q, samples_q, num_samples, 4);
  
  //hipProfilerStart();
  int threads_per_block = 8;
  int blocks_per_grid = ceil(num_samples/(1.0*threads_per_block));
  score_samples<<<blocks_per_grid, threads_per_block>>>(cu_scores, cu_samples_x, cu_samples_q, num_samples, *cu_model, *cu_obs, *cu_params, score_round, 
							cu_xi, cu_yi, cu_idx, cu_cloud, cu_normals, cu_lab, cu_vis_prob, cu_vis_pmf, cu_rands, cu_noise_models,
							cu_idx_edge, cu_P, cu_V_edge, cu_V2_edge, cu_occ_edges, cu_vis_prob_edge, cu_vis_pmf_edge);
  hipDeviceSynchronize();
  //hipProfilerStop();
  printf("scoring: %.2f ms\n", get_time_ms() - t0);
  hipError_t err;
  err = hipMemcpy(scores, cu_scores, num_samples * sizeof(double), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("scores error: %s\n", hipGetErrorString(err));
  }      
  
  free_matrix2(samples_x);
  free_matrix2(samples_q);  

  if (hipFree(cu_xi)) {
    printf("free xi\n");
  }      
  if (hipFree(cu_yi)  != hipSuccess) {
    printf("free yi\n");
  }      
  if (hipFree(cu_idx) != hipSuccess) {
    printf("free idx\n");
  }      
  if (hipFree(cu_cloud) != hipSuccess) {
    printf("free cloud\n");
  }      
  if (hipFree(cu_normals) != hipSuccess) {
    printf("free normals\n");
  }      
  if (hipFree(cu_lab) != hipSuccess) {
    printf("free lab\n");
  }
  if (hipFree(cu_rands) != hipSuccess) {
    printf("free rands\n");
  }      
  if (hipFree(cu_vis_pmf) != hipSuccess) {
    printf("free vis_pmf\n");
  }      
  if (hipFree(cu_vis_prob) != hipSuccess) {
    printf("free vis_prob\n");
  }      
  if (hipFree(cu_noise_models) != hipSuccess) {
    printf("free noise_models\n");
  }
  if (hipFree(cu_idx_edge) != hipSuccess) {
    printf("free idx_edge\n");
  }
  cu_free(cu_P, "free P\n");
  cu_free(cu_V_edge, "free V\n");
  cu_free(cu_V2_edge, "free V2\n");
  cu_free(cu_occ_edges, "free occ_edges\n");
  cu_free(cu_vis_prob_edge, "free_vis_prob_edge\n");
  cu_free(cu_vis_pmf_edge, "free vis_pmf_edge\n");
}

void cu_init() {
  hipError_t err = hipInit(0);
  //if (err != 0) 
  printf("Init error: %d\n", err);
}

void cu_init_scoring(scope_model_data_t *model_data, scope_obs_data_t *obs_data, cu_model_data_t *cu_model, cu_obs_data_t *cu_obs) {

  // Allocate all the memory
  copy_double_matrix_to_gpu(&(cu_model->points), model_data->pcd_model->points, model_data->pcd_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->normals), model_data->pcd_model->normals, model_data->pcd_model->num_points, 3);
  copy_double_arr_to_gpu(&(cu_model->normalvar), model_data->pcd_model->normalvar, model_data->pcd_model->num_points);
  copy_double_matrix_to_gpu(&(cu_model->lab), model_data->pcd_model->lab, model_data->pcd_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->ved), model_data->pcd_model->ved, model_data->pcd_model->num_points, 66);
  copy_double_matrix_to_gpu(&(cu_model->color_avg_cov), model_data->color_model->avg_cov, 3, 3);
  copy_int_arr_to_gpu(&(cu_model->color_cnts1), model_data->color_model->cnts[0], model_data->color_model->num_points);
  copy_int_arr_to_gpu(&(cu_model->color_cnts2), model_data->color_model->cnts[1], model_data->color_model->num_points);
  copy_double_matrix_to_gpu(&(cu_model->color_means1), model_data->color_model->means[0], model_data->color_model->num_points, 3);
  copy_double_matrix_to_gpu(&(cu_model->color_means2), model_data->color_model->means[1], model_data->color_model->num_points, 3);
  copy_double_matrix3d_to_gpu(&(cu_model->color_cov1), model_data->color_model->covs[0], model_data->color_model->num_points, 3, 3);
  copy_double_matrix3d_to_gpu(&(cu_model->color_cov2), model_data->color_model->covs[1], model_data->color_model->num_points, 3, 3);
  //copy_double_matrix_to_gpu(&(cu_model->fpfh_shapes), model_data->fpfh_model->shapes, model_data->fpfh_model->shape_length, 33);
  copy_double_matrix_to_gpu(&(cu_model->range_edges_model_views), model_data->range_edges_model->views, model_data->range_edges_model->num_views, 3);
  copy_int_arr_to_gpu(&(cu_model->range_edges_view_idx), model_data->range_edges_model->view_idx, model_data->range_edges_model->num_views);
  copy_int_arr_to_gpu(&(cu_model->range_edges_view_cnt), model_data->range_edges_model->view_cnt, model_data->range_edges_model->num_views);
  copy_double_matrix_to_gpu(&(cu_model->range_edges_points), model_data->range_edges_model->pcd->points, model_data->range_edges_model->pcd->num_points, 3);
  
  cu_model->num_points = model_data->pcd_model->num_points;
  cu_model->num_views = model_data->range_edges_model->num_views;
  int n_edge = arr_max_i(model_data->range_edges_model->view_cnt, model_data->range_edges_model->num_views);
  cu_model->max_num_edges = n_edge;
  // CONTINUE HERE FOR MODEL DATA COPYING ****************************

  copy_double_matrix_to_gpu(&(cu_obs->range_image), obs_data->obs_range_image->image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_cnt), obs_data->obs_range_image->cnt, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_points), obs_data->obs_range_image->points, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_normals), obs_data->obs_range_image->normals, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_idx), obs_data->obs_range_image->idx, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix_to_gpu(&(cu_obs->range_image_pcd_obs_bg_lab), obs_data->pcd_obs_bg->lab, obs_data->pcd_obs_bg->num_points, 3);
  //copy_double_matrix_to_gpu(&(cu_obs->pcd_obs_fpfh), obs_data->pcd_obs->fpfh, obs_data->pcd_obs->fpfh_length, 33);
  copy_double_matrix_to_gpu(&(cu_obs->edge_image), obs_data->obs_edge_image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);

  cu_obs->range_image_data.res = obs_data->obs_range_image->res;
  cu_obs->range_image_data.min0 = obs_data->obs_range_image->min[0];
  cu_obs->range_image_data.min1 = obs_data->obs_range_image->min[1];
  cu_obs->range_image_data.w = obs_data->obs_range_image->w;
  cu_obs->range_image_data.h = obs_data->obs_range_image->h;

  // CONTINUE HERE FOR OBS DATA COPYING ********************************

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  //hipDeviceSynchronize();
}

void cu_init_scoring_mope(scope_model_data_t model_data[], scope_obs_data_t *obs_data, int num_models, cu_model_data_t cu_model[], cu_obs_data_t *cu_obs) {
  
  // Allocate all the memory

  for (int i = 0; i < num_models; ++i) {
    copy_double_matrix_to_gpu(&(cu_model[i].points), model_data[i].pcd_model->points, model_data[i].pcd_model->num_points, 3);
    copy_double_matrix_to_gpu(&(cu_model[i].normals), model_data[i].pcd_model->normals, model_data[i].pcd_model->num_points, 3);
    copy_double_arr_to_gpu(&(cu_model[i].normalvar), model_data[i].pcd_model->normalvar, model_data[i].pcd_model->num_points);
    copy_double_matrix_to_gpu(&(cu_model[i].lab), model_data[i].pcd_model->lab, model_data[i].pcd_model->num_points, 3);
    copy_double_matrix_to_gpu(&(cu_model[i].ved), model_data[i].pcd_model->ved, model_data[i].pcd_model->num_points, 66);
    copy_double_matrix_to_gpu(&(cu_model[i].color_avg_cov), model_data[i].color_model->avg_cov, 3, 3);
    copy_int_arr_to_gpu(&(cu_model[i].color_cnts1), model_data[i].color_model->cnts[0], model_data[i].color_model->num_points);
    copy_int_arr_to_gpu(&(cu_model[i].color_cnts2), model_data[i].color_model->cnts[1], model_data[i].color_model->num_points);
    copy_double_matrix_to_gpu(&(cu_model[i].color_means1), model_data[i].color_model->means[0], model_data[i].color_model->num_points, 3);
    copy_double_matrix_to_gpu(&(cu_model[i].color_means2), model_data[i].color_model->means[1], model_data[i].color_model->num_points, 3);
    copy_double_matrix3d_to_gpu(&(cu_model[i].color_cov1), model_data[i].color_model->covs[0], model_data[i].color_model->num_points, 3, 3);
    copy_double_matrix3d_to_gpu(&(cu_model[i].color_cov2), model_data[i].color_model->covs[1], model_data[i].color_model->num_points, 3, 3);
    //copy_double_matrix_to_gpu(&(cu_model->fpfh_shapes), model_data->fpfh_model->shapes, model_data->fpfh_model->shape_length, 33);
    copy_double_matrix_to_gpu(&(cu_model[i].range_edges_model_views), model_data[i].range_edges_model->views, model_data[i].range_edges_model->num_views, 3);
    copy_int_arr_to_gpu(&(cu_model[i].range_edges_view_idx), model_data[i].range_edges_model->view_idx, model_data[i].range_edges_model->num_views);
    copy_int_arr_to_gpu(&(cu_model[i].range_edges_view_cnt), model_data[i].range_edges_model->view_cnt, model_data[i].range_edges_model->num_views);
    copy_double_matrix_to_gpu(&(cu_model[i].range_edges_points), model_data[i].range_edges_model->pcd->points, model_data[i].range_edges_model->pcd->num_points, 3);
  
    cu_model[i].num_points = model_data[i].pcd_model->num_points;
    cu_model[i].num_views = model_data[i].range_edges_model->num_views;
    int n_edge = arr_max_i(model_data[i].range_edges_model->view_cnt, model_data[i].range_edges_model->num_views);
    cu_model[i].max_num_edges = n_edge;
    // CONTINUE HERE FOR MODEL DATA COPYING ****************************
  }

  copy_double_matrix_to_gpu(&(cu_obs->range_image), obs_data->obs_range_image->image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_cnt), obs_data->obs_range_image->cnt, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_points), obs_data->obs_range_image->points, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  copy_double_matrix3d_to_gpu(&(cu_obs->range_image_normals), obs_data->obs_range_image->normals, obs_data->obs_range_image->w, obs_data->obs_range_image->h, 3);
  copy_int_matrix_to_gpu(&(cu_obs->range_image_idx), obs_data->obs_range_image->idx, obs_data->obs_range_image->w, obs_data->obs_range_image->h);
  copy_double_matrix_to_gpu(&(cu_obs->range_image_pcd_obs_bg_lab), obs_data->pcd_obs_bg->lab, obs_data->pcd_obs_bg->num_points, 3);
  //copy_double_matrix_to_gpu(&(cu_obs->pcd_obs_fpfh), obs_data->pcd_obs->fpfh, obs_data->pcd_obs->fpfh_length, 33);
  copy_double_matrix_to_gpu(&(cu_obs->edge_image), obs_data->obs_edge_image, obs_data->obs_range_image->w, obs_data->obs_range_image->h);

  cu_obs->range_image_data.res = obs_data->obs_range_image->res;
  cu_obs->range_image_data.min0 = obs_data->obs_range_image->min[0];
  cu_obs->range_image_data.min1 = obs_data->obs_range_image->min[1];
  cu_obs->range_image_data.w = obs_data->obs_range_image->w;
  cu_obs->range_image_data.h = obs_data->obs_range_image->h;

  // CONTINUE HERE FOR OBS DATA COPYING ********************************

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  //hipDeviceSynchronize();
}

void cu_free_all_the_things(cu_model_data_t *cu_model, cu_obs_data_t *cu_obs) {
  // Free ALL the things!!!

  hipFree(cu_model->points.ptr);
  hipFree(cu_model->normals.ptr);
  hipFree(cu_model->normalvar.ptr);
  hipFree(cu_model->lab.ptr);
  hipFree(cu_model->ved.ptr);
  hipFree(cu_model->color_avg_cov.ptr);
  hipFree(cu_model->color_means1.ptr);
  hipFree(cu_model->color_means2.ptr);
  //hipFree(cu_model->fpfh_shapes.ptr);
  hipFree(cu_model->range_edges_model_views.ptr);
  hipFree(cu_model->range_edges_points.ptr);
  hipFree(cu_model->color_cov1.ptr);
  hipFree(cu_model->color_cov2.ptr);
  hipFree(cu_model->color_cnts1.ptr);
  hipFree(cu_model->color_cnts2.ptr);
  hipFree(cu_model->range_edges_view_idx.ptr);
  hipFree(cu_model->range_edges_view_cnt.ptr);
  
  hipFree(cu_obs->range_image.ptr);
  hipFree(cu_obs->range_image_idx.ptr);
  hipFree(cu_obs->range_image_pcd_obs_bg_lab.ptr);
  //hipFree(cu_obs->pcd_obs_fpfh.ptr);
  hipFree(cu_obs->edge_image.ptr);
  hipFree(cu_obs->range_image_points.ptr);
  hipFree(cu_obs->range_image_normals.ptr);
  hipFree(cu_obs->range_image_cnt.ptr);
 
  hiprandDestroyGenerator(gen);
}

void cu_free_all_the_things_mope(cu_model_data_t cu_model[], cu_obs_data_t *cu_obs, int num_models) {
  // Free ALL the things!!!
  
  for (int i = 0; i < num_models; ++i) {
    hipFree(cu_model[i].points.ptr);
    hipFree(cu_model[i].normals.ptr);
    hipFree(cu_model[i].normalvar.ptr);
    hipFree(cu_model[i].lab.ptr);
    hipFree(cu_model[i].ved.ptr);
    hipFree(cu_model[i].color_avg_cov.ptr);
    hipFree(cu_model[i].color_means1.ptr);
    hipFree(cu_model[i].color_means2.ptr);
    //hipFree(cu_model->fpfh_shapes.ptr);
    hipFree(cu_model[i].range_edges_model_views.ptr);
    hipFree(cu_model[i].range_edges_points.ptr);
    hipFree(cu_model[i].color_cov1.ptr);
    hipFree(cu_model[i].color_cov2.ptr);
    hipFree(cu_model[i].color_cnts1.ptr);
    hipFree(cu_model[i].color_cnts2.ptr);
    hipFree(cu_model[i].range_edges_view_idx.ptr);
    hipFree(cu_model[i].range_edges_view_cnt.ptr);
  }
  hipFree(cu_obs->range_image.ptr);
  hipFree(cu_obs->range_image_idx.ptr);
  hipFree(cu_obs->range_image_pcd_obs_bg_lab.ptr);
  //hipFree(cu_obs->pcd_obs_fpfh.ptr);
  hipFree(cu_obs->edge_image.ptr);
  hipFree(cu_obs->range_image_points.ptr);
  hipFree(cu_obs->range_image_normals.ptr);
  hipFree(cu_obs->range_image_cnt.ptr);
 
  hiprandDestroyGenerator(gen);
}

